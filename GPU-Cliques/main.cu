#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <set>
#include <iostream>
#include <fstream>
#include <string>
#include <cstdio>
#include <sstream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <sm_30_intrinsics.h>
#include <device_atomic_functions.h>
using namespace std;

// GPU: GTX 1660 Super
// SM's: 22
// Threads per SM: 1024
// Global Memory: 6 GB
// Shared Memory: 48 KB

// global memory size: 1.500.000.000 ints
#define TASKS_SIZE 15000000
#define EXPAND_THRESHOLD 286
#define BUFFER_SIZE 100000000
#define BUFFER_OFFSET_SIZE 1000000
#define CLIQUES_SIZE 50000000
#define CLIQUES_OFFSET_SIZE 500000
#define CLIQUES_PERCENT 50

// per warp
#define WCLIQUES_SIZE 50000
#define WCLIQUES_OFFSET_SIZE 500
#define WTASKS_SIZE 1000000
#define WTASKS_OFFSET_SIZE 5000
#define WVERTICES_SIZE 40000

// shared memory size: 12.300 ints
#define VERTICES_SIZE 110
 
#define BLOCK_SIZE 416
#define NUM_OF_BLOCKS 22
#define WARP_SIZE 32

// VERTEX DATA
struct Vertex
{
    int vertexid;
    int label;
    int indeg;
    int exdeg;
    int lvl2adj;
};

// CPU GRAPH / CONSTRUCTOR
class CPU_Graph
{
    public:

    int number_of_vertices;
    int number_of_edges;

    // one dimentional arrays of 1hop and 2hop neighbors and the offsets for each vertex
    int* onehop_neighbors;
    uint64_t* onehop_offsets;
    int* twohop_neighbors;
    uint64_t* twohop_offsets;

    uint64_t number_of_onehop_neighbors;
    uint64_t number_of_twohop_neighbors;

    CPU_Graph::CPU_Graph(ifstream& graph_stream)
    {
        // used to allocate memory for neighbor arrays
        number_of_onehop_neighbors = 0;
        number_of_twohop_neighbors = 0;

        // vectors of sets of 1hop and 2hop neighbors
        vector<set<int>> onehop_neighbors_vector;
        vector<set<int>> twohop_neighbors_vector;

        // generate 1hop neighbors vector
        string line;
        string neighbor;
        while (graph_stream.good()) {
            getline(graph_stream, line);
            if (line.length() != 0) {
                stringstream neighbor_stream(line);
                set<int> tempset;
                while (!neighbor_stream.eof()) {
                    getline(neighbor_stream, neighbor, ' ');
                    int neighbor_id;
                    try {
                        neighbor_id = stoi(neighbor);
                        tempset.insert(neighbor_id);
                        number_of_onehop_neighbors++;
                    }
                    catch (const std::invalid_argument& e) {}
                }
                onehop_neighbors_vector.push_back(tempset);
            }
            else {
                set<int> tempset;
                onehop_neighbors_vector.push_back(tempset);
            }
        }

        // set V and E
        number_of_vertices = onehop_neighbors_vector.size();
        number_of_edges = number_of_onehop_neighbors / 2;

        // generate 2hop neighbors vector
        int current_vertex = 0;
        for (set<int> vertex_neighbors : onehop_neighbors_vector) {
            set<int> tempset(vertex_neighbors);
            for (int neighbor : vertex_neighbors) {
                for (int twohop_neighbor : onehop_neighbors_vector.at(neighbor)) {
                    if (twohop_neighbor != current_vertex) {
                        tempset.insert(twohop_neighbor);
                    }
                }
            }
            twohop_neighbors_vector.push_back(tempset);
            number_of_twohop_neighbors += tempset.size();
            current_vertex++;
        }

        // convert onehop vector to arrays
        onehop_neighbors = new int[number_of_onehop_neighbors];
        onehop_offsets = new uint64_t[number_of_vertices + 1];
        if (onehop_neighbors == nullptr || onehop_offsets == nullptr) {
            cout << "ERROR: bad malloc" << endl;
        }
        onehop_offsets[0] = 0;
        int offset = 0;
        for (int i = 0; i < onehop_neighbors_vector.size(); i++) {
            offset += onehop_neighbors_vector.at(i).size();
            onehop_offsets[i + 1] = offset;
            int j = 0;
            for (int neighbor : onehop_neighbors_vector.at(i)) {
                onehop_neighbors[onehop_offsets[i] + j] = neighbor;
                j++;
            }
        }

        //convert twohop vector to arrays
        twohop_neighbors = new int[number_of_twohop_neighbors];
        twohop_offsets = new uint64_t[number_of_vertices + 1];
        if (twohop_neighbors == nullptr || twohop_offsets == nullptr) {
            cout << "ERROR: bad malloc" << endl;
        }
        twohop_offsets[0] = 0;
        offset = 0;
        for (int i = 0; i < twohop_neighbors_vector.size(); i++) {
            offset += twohop_neighbors_vector.at(i).size();
            twohop_offsets[i + 1] = offset;
            int j = 0;
            for (int neighbor : twohop_neighbors_vector.at(i)) {
                twohop_neighbors[twohop_offsets[i] + j] = neighbor;
                j++;
            }
        }
    }

    CPU_Graph::~CPU_Graph() {
        delete onehop_neighbors;
        delete onehop_offsets;
        delete twohop_neighbors;
        delete twohop_offsets;
    }
};

// CPU DATA
struct CPU_Data
{
    uint64_t* tasks1_count;
    uint64_t* tasks1_offset;
    Vertex* tasks1_vertices;

    uint64_t* buffer_count;
    uint64_t* buffer_offset;
    Vertex* buffer_vertices;

    bool* maximal_expansion;
    bool* dumping_cliques;
};

// CPU CLIQUES
struct CPU_Cliques
{
    uint64_t* cliques_count;
    uint64_t* cliques_offset;
    int* cliques_vertex;
};

// DEVICE DATA
struct GPU_Data
{
    // GPU DATA
    uint64_t* current_level;

    uint64_t* tasks1_count;
    uint64_t* tasks1_offset;
    Vertex* tasks1_vertices;

    uint64_t* tasks2_count;
    uint64_t* tasks2_offset;
    Vertex* tasks2_vertices;

    uint64_t* buffer_count;
    uint64_t* buffer_offset;
    Vertex* buffer_vertices;

    uint64_t* wtasks_count;
    uint64_t* wtasks_offset;
    Vertex* wtasks_vertices;

    Vertex* wvertices;

    int* total_tasks;

    bool* maximal_expansion;
    bool* dumping_cliques;

    double* minimum_degree_ratio;
    int* minimum_degrees;
    int* minimum_clique_size;

    uint64_t* buffer_offset_start;
    uint64_t* buffer_start;
    uint64_t* cliques_offset_start;
    uint64_t* cliques_start;

    // DEBUG
    bool* debug;
    int* idebug;

    // GPU GRAPH
    int* number_of_vertices;
    int* number_of_edges;

    int* onehop_neighbors;
    uint64_t* onehop_offsets;
    int* twohop_neighbors;
    uint64_t* twohop_offsets;

    // GPU CLIQUES
    uint64_t* cliques_count;
    uint64_t* cliques_offset;
    int* cliques_vertex;

    uint64_t* wcliques_count;
    uint64_t* wcliques_offset;
    int* wcliques_vertex;

    int* total_cliques;
};

// WARP DATA
struct Warp_Data
{
    uint64_t start[(BLOCK_SIZE / WARP_SIZE)];
    uint64_t end[(BLOCK_SIZE / WARP_SIZE)];
    int tot_vert[(BLOCK_SIZE / WARP_SIZE)];
    int num_mem[(BLOCK_SIZE / WARP_SIZE)];
    int num_cand[(BLOCK_SIZE / WARP_SIZE)];
    int expansions[(BLOCK_SIZE / WARP_SIZE)];

    int number_of_members[(BLOCK_SIZE / WARP_SIZE)];
    int number_of_candidates[(BLOCK_SIZE / WARP_SIZE)];
    int total_vertices[(BLOCK_SIZE / WARP_SIZE)];

    Vertex shared_vertices[VERTICES_SIZE * (BLOCK_SIZE / WARP_SIZE)];

    int minimum_external_degree[(BLOCK_SIZE / WARP_SIZE)];
    int Lower_bound[(BLOCK_SIZE / WARP_SIZE)];
    int Upper_bound[(BLOCK_SIZE / WARP_SIZE)];

    int tightened_Upper_bound[(BLOCK_SIZE / WARP_SIZE)];
    int min_clq_indeg[(BLOCK_SIZE / WARP_SIZE)];
    int min_indeg_exdeg[(BLOCK_SIZE / WARP_SIZE)];
    int min_clq_totaldeg[(BLOCK_SIZE / WARP_SIZE)];
    int sum_clq_indeg[(BLOCK_SIZE / WARP_SIZE)];
    int sum_candidate_indeg[(BLOCK_SIZE / WARP_SIZE)];

    bool invalid_bounds[(BLOCK_SIZE / WARP_SIZE)];
    bool failed_found[(BLOCK_SIZE / WARP_SIZE)];
};

// LOCAL DATA
struct Local_Data
{
    Vertex* read_vertices;
    uint64_t* read_offsets;
    uint64_t* read_count;

    Vertex* vertices;
    int idx;
    int warp_in_block_idx;
};

// METHODS
void calculate_minimum_degrees(CPU_Graph& graph);
void search(CPU_Graph& input_graph, ofstream& temp_results);
void allocate_memory(CPU_Data& host_data, GPU_Data& dd, CPU_Cliques& host_cliques, CPU_Graph& input_graph);
void initialize_tasks(CPU_Graph& graph, CPU_Data& host_data);
void move_to_gpu(CPU_Data& host_data, GPU_Data& dd);
void dump_cliques(CPU_Cliques& host_cliques, GPU_Data& dd, ofstream& output_file);
void free_memory(CPU_Data& host_data, GPU_Data& dd, CPU_Cliques& host_cliques);
void RemoveNonMax(char* szset_filename, char* szoutput_filename);

int binary_search_array(int* search_array, int array_size, int search_number);
int sort_vertices(const void* a, const void* b);
inline int get_mindeg(int clique_size);
inline bool cand_isvalid(Vertex& vertex, int clique_size);
inline void chkerr(hipError_t code);

void print_CPU_Data(CPU_Data& host_data);
void print_GPU_Data(GPU_Data& dd);
void print_CPU_Graph(CPU_Graph& host_graph);
void print_GPU_Graph(GPU_Data& dd, CPU_Graph& host_graph);
void print_WTask_Buffers(GPU_Data& dd);
void print_WClique_Buffers(GPU_Data& dd);
void print_GPU_Cliques(GPU_Data& dd);
void print_CPU_Cliques(CPU_Cliques& host_cliques);
void print_Data_Sizes(GPU_Data& dd);
void print_vertices(Vertex* vertices, int size);
void print_Data_Sizes_Every(GPU_Data& dd, int every);
void print_Warp_Data_Sizes(GPU_Data& dd);
void print_All_Warp_Data_Sizes(GPU_Data& dd);
void print_Warp_Data_Sizes_Every(GPU_Data& dd, int every);
void print_All_Warp_Data_Sizes_Every(GPU_Data& dd, int every);
void print_debug(GPU_Data& dd);
void print_idebug(GPU_Data& dd);
void print_idebug(GPU_Data& dd);

// KERNELS
__global__ void expand_level(GPU_Data dd);
__global__ void transfer_buffers(GPU_Data dd);
__global__ void fill_from_buffer(GPU_Data dd);
__device__ int lookahead_pruning(GPU_Data& dd, Warp_Data& wd, Local_Data& ld);
__device__ int remove_one_vertex(GPU_Data& dd, Warp_Data& wd, Local_Data& ld);
__device__ int add_one_vertex(GPU_Data& dd, Warp_Data& wd, Local_Data& ld);
__device__ void check_for_clique(GPU_Data& dd, Warp_Data& wd, Local_Data& ld);
__device__ void write_to_tasks(GPU_Data& dd, Warp_Data& wd, Local_Data& ld);
__device__ void diameter_pruning(GPU_Data& dd, Warp_Data& wd, Local_Data& ld, int pvertexid);
__device__ void degree_pruning(GPU_Data& dd, Warp_Data& wd, Local_Data& ld, bool& failed_found);
__device__ void update_degrees(GPU_Data& dd, Warp_Data& wd, Local_Data& ld, int number_of_removed_candidates);
__device__ void calculate_LU_bounds(GPU_Data& dd, Warp_Data& wd, Local_Data& ld);

__device__ void degree_pruning_nonLU(GPU_Data& dd, Warp_Data& wd, Local_Data& ld, bool& failed_found);
__device__ void device_sort(Vertex* target, int size, int lane_idx);
__device__ __forceinline int sort_vert(Vertex& vertex1, Vertex& vertex2);
__device__ int device_bsearch_array(int* search_array, int array_size, int search_number);
__device__ __forceinline bool device_cand_isvalid(Vertex& vertex, int number_of_members, GPU_Data& dd);
__device__ __forceinline bool device_cand_isvalid_LU(Vertex& vertex, GPU_Data& dd, Warp_Data& wd, Local_Data& ld);
__device__ __forceinline bool device_vert_isextendable(Vertex& vertex, int number_of_members, GPU_Data& dd);
__device__ __forceinline bool device_vert_isextendable_LU(Vertex& vertex, GPU_Data& dd, Warp_Data& wd, Local_Data& ld);
__device__ __forceinline int device_get_mindeg(int number_of_members, GPU_Data& dd);

// TODO - make local
// INPUT SETTINGS
double minimum_degree_ratio;
int minimum_clique_size;
int* minimum_degrees;



// TODO - test program on larger graphs
// TODO - increase thread usage by monitoring and improving memory usage
// TODO - test if it would be beneficial to coalesce memory access in for loops throughout the program, check out cuts writing on this

// MAIN
int main(int argc, char* argv[])
{
    // ENSURE PROPER USAGE
    if (argc != 5) {
        printf("Usage: ./main <graph_file> <gamma> <min_size> <output_file.txt>\n");
        return 1;
    }
    ifstream graph_stream(argv[1], ios::in);
    if (!graph_stream.is_open()) {
        printf("invalid graph file\n");
        return 1;
    }
    minimum_degree_ratio = atof(argv[2]);
    if (minimum_degree_ratio < .5 || minimum_degree_ratio>1) {
        printf("minimum degree ratio must be between .5 and 1 inclusive\n");
        return 1;
    }
    minimum_clique_size = atoi(argv[3]);
    if (minimum_clique_size <= 1) {
        printf("minimum size must be greater than 1\n");
        return 1;
    }

    // TIME
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // GRAPH / MINDEGS
    cout << ">:PRE-PROCESSING" << endl;
    CPU_Graph input_graph(graph_stream);
    graph_stream.close();
    calculate_minimum_degrees(input_graph);
    ofstream temp_results("temp.txt");

    // DEBUG
    //print_CPU_Graph(input_graph);

    // SEARCH
    search(input_graph, temp_results);

    temp_results.close();

    // RM NON-MAX
    RemoveNonMax("temp.txt", argv[4]);

    // Record the stop event
    hipEventRecord(stop);

    // TIME
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << ">:TIME: " << milliseconds << " ms" << endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout << ">:PROGRAM END" << endl;
    return 0;
}



// --- HOST METHODS --- 

// initializes minimum degrees array
void calculate_minimum_degrees(CPU_Graph& graph)
{
    minimum_degrees = new int[graph.number_of_vertices + 1];
    minimum_degrees[0] = 0;
    for (int i = 1; i <= graph.number_of_vertices; i++) {
        minimum_degrees[i] = ceil(minimum_degree_ratio * (i - 1));
    }
}

void search(CPU_Graph& input_graph, ofstream& temp_results) 
{
    // DATA STRUCTURES
    CPU_Data host_data;
    CPU_Cliques host_cliques;
    GPU_Data dd;

    // HANDLE MEMORY
    allocate_memory(host_data, dd, host_cliques, input_graph);
    hipDeviceSynchronize();

    // INITIALIZE TASKS
    cout << ">:INITIALIZING TASKS" << endl;
    initialize_tasks(input_graph, host_data);

    // TRANSFER TO GPU
    move_to_gpu(host_data, dd);
    hipDeviceSynchronize();

    // DEBUG
    //print_GPU_Graph(dd, input_graph);
    //print_CPU_Data(host_data);
    //print_GPU_Data(dd);
    print_Data_Sizes(dd);

    // TODO - check cuts for hipDeviceSynchronize
    // EXPAND LEVEL
    cout << ">:BEGINNING EXPANSION" << endl;
    while (!(*host_data.maximal_expansion))
    {
        // reset loop variables
        chkerr(hipMemset(dd.maximal_expansion, true, sizeof(bool)));
        chkerr(hipMemset(dd.dumping_cliques, false, sizeof(bool)));
        hipDeviceSynchronize();

        // expand all tasks in 'tasks' array, each warp will write to their respective warp tasks buffer in global memory
        expand_level<<<NUM_OF_BLOCKS, BLOCK_SIZE>>>(dd);
        hipDeviceSynchronize();

        // DEBUG
        //print_WClique_Buffers(dd);
        //print_WTask_Buffers(dd);
        print_Warp_Data_Sizes_Every(dd, 1);
        //print_All_Warp_Data_Sizes_Every(dd, 1);

        // consolidate all the warp tasks/cliques buffers into the next global tasks array, buffer, and cliques
        transfer_buffers<<<NUM_OF_BLOCKS, BLOCK_SIZE>>>(dd);
        hipDeviceSynchronize();

        // if not enough tasks were generated when expanding the previous level to fill the next tasks array the program will attempt to fill the tasks array by popping tasks from the buffer
        fill_from_buffer<<<NUM_OF_BLOCKS, BLOCK_SIZE>>>(dd);
        hipDeviceSynchronize();

        // update the loop variables
        chkerr(hipMemcpy(host_data.maximal_expansion, dd.maximal_expansion, sizeof(bool), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(host_data.dumping_cliques, dd.dumping_cliques, sizeof(bool), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();

        if (*host_data.dumping_cliques) {
            dump_cliques(host_cliques, dd, temp_results);
        }

        // DEBUG
        //print_GPU_Data(dd);
        //print_GPU_Cliques(dd);
        print_Data_Sizes_Every(dd, 1);
        //print_debug(dd);
        //print_idebug(dd);
    }

    dump_cliques(host_cliques, dd, temp_results);

    // FREE MEMORY
    free_memory(host_data, dd, host_cliques);
}

// allocates memory for the data structures on the host and device
void allocate_memory(CPU_Data& host_data, GPU_Data& dd, CPU_Cliques& host_cliques, CPU_Graph& input_graph)
{
    int number_of_warps = (NUM_OF_BLOCKS * BLOCK_SIZE) / WARP_SIZE;

    // GPU GRAPH
    chkerr(hipMalloc((void**)&dd.number_of_vertices, sizeof(int)));
    chkerr(hipMalloc((void**)&dd.number_of_edges, sizeof(int)));
    chkerr(hipMalloc((void**)&dd.onehop_neighbors, sizeof(int) * input_graph.number_of_onehop_neighbors));
    chkerr(hipMalloc((void**)&dd.onehop_offsets, sizeof(uint64_t) * (input_graph.number_of_vertices + 1)));
    chkerr(hipMalloc((void**)&dd.twohop_neighbors, sizeof(int) * input_graph.number_of_twohop_neighbors));
    chkerr(hipMalloc((void**)&dd.twohop_offsets, sizeof(uint64_t) * (input_graph.number_of_vertices + 1)));

    chkerr(hipMemcpy(dd.number_of_vertices, &(input_graph.number_of_vertices), sizeof(int), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(dd.number_of_edges, &(input_graph.number_of_edges), sizeof(int), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(dd.onehop_neighbors, input_graph.onehop_neighbors, sizeof(int) * input_graph.number_of_onehop_neighbors, hipMemcpyHostToDevice));
    chkerr(hipMemcpy(dd.onehop_offsets, input_graph.onehop_offsets, sizeof(uint64_t) * (input_graph.number_of_vertices + 1), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(dd.twohop_neighbors, input_graph.twohop_neighbors, sizeof(int) * input_graph.number_of_twohop_neighbors, hipMemcpyHostToDevice));
    chkerr(hipMemcpy(dd.twohop_offsets, input_graph.twohop_offsets, sizeof(uint64_t) * (input_graph.number_of_vertices + 1), hipMemcpyHostToDevice));

    // CPU DATA
    host_data.tasks1_count = new uint64_t;
    host_data.tasks1_offset = new uint64_t[EXPAND_THRESHOLD + 1];
    host_data.tasks1_vertices = new Vertex[TASKS_SIZE];

    host_data.tasks1_offset[0] = 0;
    (*(host_data.tasks1_count)) = 0;

    host_data.buffer_count = new uint64_t;
    host_data.buffer_offset = new uint64_t[BUFFER_OFFSET_SIZE];
    host_data.buffer_vertices = new Vertex[BUFFER_SIZE];

    host_data.buffer_offset[0] = 0;
    (*(host_data.buffer_count)) = 0;

    host_data.maximal_expansion = new bool;
    host_data.dumping_cliques = new bool;

    (*host_data.maximal_expansion) = false;
    (*host_data.dumping_cliques) = false;

    // GPU DATA
    chkerr(hipMalloc((void**)&dd.current_level, sizeof(uint64_t)));

    uint64_t temp = 1;
    chkerr(hipMemcpy(dd.current_level, &temp, sizeof(uint64_t), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&dd.tasks1_count, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&dd.tasks1_offset, sizeof(uint64_t) * (EXPAND_THRESHOLD + 1)));
    chkerr(hipMalloc((void**)&dd.tasks1_vertices, sizeof(Vertex) * TASKS_SIZE));

    chkerr(hipMemset(dd.tasks1_offset, 0, sizeof(uint64_t)));
    chkerr(hipMemset(dd.tasks1_count, 0, sizeof(uint64_t)));

    chkerr(hipMalloc((void**)&dd.tasks2_count, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&dd.tasks2_offset, sizeof(uint64_t) * (EXPAND_THRESHOLD + 1)));
    chkerr(hipMalloc((void**)&dd.tasks2_vertices, sizeof(Vertex) * TASKS_SIZE));

    chkerr(hipMemset(dd.tasks2_offset, 0, sizeof(uint64_t)));
    chkerr(hipMemset(dd.tasks2_count, 0, sizeof(uint64_t)));

    chkerr(hipMalloc((void**)&dd.buffer_count, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&dd.buffer_offset, sizeof(uint64_t) * BUFFER_OFFSET_SIZE));
    chkerr(hipMalloc((void**)&dd.buffer_vertices, sizeof(Vertex) * BUFFER_SIZE));

    chkerr(hipMemset(dd.buffer_offset, 0, sizeof(uint64_t)));
    chkerr(hipMemset(dd.buffer_count, 0, sizeof(uint64_t)));

    chkerr(hipMalloc((void**)&dd.wtasks_count, sizeof(uint64_t) * number_of_warps));
    chkerr(hipMalloc((void**)&dd.wtasks_offset, (sizeof(uint64_t) * WTASKS_OFFSET_SIZE) * number_of_warps));
    chkerr(hipMalloc((void**)&dd.wtasks_vertices, (sizeof(Vertex) * WTASKS_SIZE) * number_of_warps));

    chkerr(hipMemset(dd.wtasks_offset, 0, (sizeof(uint64_t) * WTASKS_OFFSET_SIZE) * number_of_warps));
    chkerr(hipMemset(dd.wtasks_count, 0, sizeof(uint64_t) * number_of_warps));

    chkerr(hipMalloc((void**)&dd.wvertices, (sizeof(Vertex) * WVERTICES_SIZE) * number_of_warps));

    chkerr(hipMalloc((void**)&dd.maximal_expansion, sizeof(bool)));
    chkerr(hipMalloc((void**)&dd.dumping_cliques, sizeof(bool)));

    chkerr(hipMemset(dd.maximal_expansion, false, sizeof(bool)));
    chkerr(hipMemset(dd.dumping_cliques, false, sizeof(bool)));

    chkerr(hipMalloc((void**)&dd.minimum_degree_ratio, sizeof(double)));
    chkerr(hipMalloc((void**)&dd.minimum_degrees, sizeof(int) * (input_graph.number_of_vertices + 1)));
    chkerr(hipMalloc((void**)&dd.minimum_clique_size, sizeof(int)));

    chkerr(hipMemcpy(dd.minimum_degree_ratio, &minimum_degree_ratio, sizeof(double), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(dd.minimum_degrees, minimum_degrees, sizeof(int) * (input_graph.number_of_vertices + 1), hipMemcpyHostToDevice));
    chkerr(hipMemcpy(dd.minimum_clique_size, &minimum_clique_size, sizeof(int), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&dd.total_tasks, sizeof(int)));

    chkerr(hipMemset(dd.total_tasks, 0, sizeof(int)));

    // CPU CLIQUES
    host_cliques.cliques_count = new uint64_t;
    host_cliques.cliques_vertex = new int[CLIQUES_SIZE];
    host_cliques.cliques_offset = new uint64_t[CLIQUES_OFFSET_SIZE];

    host_cliques.cliques_offset[0] = 0;
    (*(host_cliques.cliques_count)) = 0;

    // GPU CLIQUES
    chkerr(hipMalloc((void**)&dd.cliques_count, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&dd.cliques_vertex, sizeof(int) * CLIQUES_SIZE));
    chkerr(hipMalloc((void**)&dd.cliques_offset, sizeof(uint64_t) * CLIQUES_OFFSET_SIZE));

    chkerr(hipMemset(dd.cliques_offset, 0, sizeof(uint64_t)));
    chkerr(hipMemset(dd.cliques_count, 0, sizeof(uint64_t)));

    chkerr(hipMalloc((void**)&dd.wcliques_count, sizeof(uint64_t) * number_of_warps));
    chkerr(hipMalloc((void**)&dd.wcliques_offset, (sizeof(uint64_t)* WCLIQUES_OFFSET_SIZE)* number_of_warps));
    chkerr(hipMalloc((void**)&dd.wcliques_vertex, (sizeof(int) * WCLIQUES_SIZE) * number_of_warps));

    chkerr(hipMemset(dd.wcliques_offset, 0, (sizeof(uint64_t)* WCLIQUES_OFFSET_SIZE)* number_of_warps));
    chkerr(hipMemset(dd.wcliques_count, 0, sizeof(uint64_t)* number_of_warps));

    chkerr(hipMalloc((void**)&dd.total_cliques, sizeof(int)));

    chkerr(hipMemset(dd.total_cliques, 0, sizeof(int)));

    chkerr(hipMalloc((void**)&dd.buffer_offset_start, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&dd.buffer_start, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&dd.cliques_offset_start, sizeof(uint64_t)));
    chkerr(hipMalloc((void**)&dd.cliques_start, sizeof(uint64_t)));

    // DEBUG
    chkerr(hipMalloc((void**)&dd.debug, sizeof(bool)));
    chkerr(hipMalloc((void**)&dd.idebug, sizeof(int)));

    chkerr(hipMemset(dd.debug, false, sizeof(bool)));
    chkerr(hipMemset(dd.idebug, 0, sizeof(int)));
}

// processes 0th and 1st level of expansion
void initialize_tasks(CPU_Graph& graph, CPU_Data& host_data)
{
    // intersection
    int pvertexid;
    uint64_t pneighbors_start;
    uint64_t pneighbors_end;
    int pneighbors_count;
    int phelper1;
    int phelper2;

    // cover pruning
    int number_of_covered_vertices;
    int maximum_degree;
    int maximum_degree_index;

    // degree pruning
    int number_of_removed_candidates;

    // vertices information
    int expansions;
    int total_vertices;
    Vertex* old_vertices;
    int total_new_vertices;
    Vertex* new_vertices;



    // initialize vertices
    total_vertices = graph.number_of_vertices;
    old_vertices = new Vertex[total_vertices];
    for (int i = 0; i < total_vertices; i++) {
        old_vertices[i].vertexid = i;
        old_vertices[i].label = 0;
        old_vertices[i].indeg = 0;
        old_vertices[i].exdeg = graph.onehop_offsets[i + 1] - graph.onehop_offsets[i];
        old_vertices[i].lvl2adj = graph.twohop_offsets[i + 1] - graph.twohop_offsets[i];
    }



    // DEGREE-BASED PRUNING
    do {
        // remove cands that do not meet the deg requirement
        number_of_removed_candidates = 0;
        for (int i = 0; i < total_vertices; i++) {
            if (!cand_isvalid(old_vertices[i], 0)) {
                old_vertices[i].label = -1;
                number_of_removed_candidates++;
            }
        }
        qsort(old_vertices, total_vertices, sizeof(Vertex), sort_vertices);

        for (int i = 0; i < total_vertices - number_of_removed_candidates; i++) {
            pvertexid = old_vertices[i].vertexid;
            for (int j = total_vertices - number_of_removed_candidates; j < total_vertices; j++) {
                phelper1 = old_vertices[j].vertexid;
                pneighbors_start = graph.onehop_offsets[phelper1];
                pneighbors_end = graph.onehop_offsets[phelper1 + 1];
                pneighbors_count = pneighbors_end - pneighbors_start;
                phelper2 = binary_search_array(graph.onehop_neighbors + pneighbors_start, pneighbors_count, pvertexid);
                if (phelper2 != -1) {
                    old_vertices[i].exdeg--;
                }

                pneighbors_start = graph.twohop_offsets[phelper1];
                pneighbors_end = graph.twohop_offsets[phelper1 + 1];
                pneighbors_count = pneighbors_end - pneighbors_start;
                phelper2 = binary_search_array(graph.twohop_neighbors + pneighbors_start, pneighbors_count, pvertexid);
                if (phelper2 != -1) {
                    old_vertices[i].lvl2adj--;
                }
            }
        }
        total_vertices -= number_of_removed_candidates;
    } while (number_of_removed_candidates > 0);
    


    // FIRST ROUND COVER PRUNING
    maximum_degree = 0;
    maximum_degree_index = 0;
    for (int i = 0; i < total_vertices; i++) {
        if (old_vertices[i].exdeg > maximum_degree) {
            maximum_degree = old_vertices[i].exdeg;
            maximum_degree_index = i;
        }
    }
    old_vertices[maximum_degree_index].label = 3;

    // set all neighbors of cover vertices as covered
    pvertexid = old_vertices[maximum_degree_index].vertexid;
    qsort(old_vertices, total_vertices, sizeof(Vertex), sort_vertices);
    number_of_covered_vertices = 0;
    for (int i = 0; i < total_vertices-1; i++) {
        phelper1 = old_vertices[i].vertexid;
        pneighbors_start = graph.onehop_offsets[phelper1];
        pneighbors_end = graph.onehop_offsets[phelper1 + 1];
        pneighbors_count = pneighbors_end - pneighbors_start;
        phelper2 = binary_search_array(graph.onehop_neighbors + pneighbors_start, pneighbors_count, pvertexid);
        if (phelper2 != -1) {
            old_vertices[i].label = 2;
            number_of_covered_vertices++;
        }
    }
    qsort(old_vertices, total_vertices, sizeof(Vertex), sort_vertices);



    // NEXT LEVEL
    expansions = total_vertices;
    for (int i = number_of_covered_vertices; i < expansions; i++)
    {



        // REMOVE CANDIDATE
        // only done after first iteration of for loop
        if (i > number_of_covered_vertices) {
            total_vertices--;

            // update info of vertices connected to removed cand
            pvertexid = old_vertices[total_vertices].vertexid;
            for (int j = 0; j < total_vertices; j++) {
                phelper1 = old_vertices[j].vertexid;
                pneighbors_start = graph.onehop_offsets[phelper1];
                pneighbors_end = graph.onehop_offsets[phelper1];
                pneighbors_count = pneighbors_end - pneighbors_start;
                phelper2 = binary_search_array(graph.onehop_neighbors + pneighbors_start, pneighbors_count, pvertexid);
                if (phelper2 != -1) {
                    old_vertices[j].exdeg--;
                }

                pneighbors_start = graph.twohop_offsets[phelper1];
                pneighbors_end = graph.twohop_offsets[phelper1];
                pneighbors_count = pneighbors_end - pneighbors_start;
                phelper2 = binary_search_array(graph.twohop_neighbors + pneighbors_start, pneighbors_count, pvertexid);
                if (phelper2 != -1) {
                    old_vertices[j].lvl2adj--;
                }
            }
        }

        // break if not enough vertices as only less will be added in the next iteration
        if (total_vertices < minimum_clique_size) {
            break;
        }



        // NEW VERTICES
        new_vertices = new Vertex[total_vertices];
        total_new_vertices = total_vertices;
        for (int j = 0; j < total_new_vertices; j++) {
            new_vertices[j].vertexid = old_vertices[j].vertexid;
            new_vertices[j].label = old_vertices[j].label;
            new_vertices[j].indeg = old_vertices[j].indeg;
            new_vertices[j].exdeg = old_vertices[j].exdeg;
            new_vertices[j].lvl2adj = old_vertices[j].lvl2adj;
        }

        // set all covered vertices from previous level as candidates
        for (int j = 0; j < number_of_covered_vertices; j++) {
            new_vertices[j].label = 0;
        }
        


        // ADD ONE VERTEX
        new_vertices[total_new_vertices - 1].label = 1;
        pvertexid = new_vertices[total_new_vertices - 1].vertexid;
        for (int j = 0; j < total_vertices; j++) {
            phelper1 = new_vertices[j].vertexid;
            pneighbors_start = graph.onehop_offsets[phelper1];
            pneighbors_end = graph.onehop_offsets[phelper1 + 1];
            pneighbors_count = pneighbors_end - pneighbors_start;
            phelper2 = binary_search_array(graph.onehop_neighbors + pneighbors_start, pneighbors_count, pvertexid);
            if (phelper2 != -1) {
                new_vertices[j].exdeg--;
                new_vertices[j].indeg++;
            }
        }
        qsort(new_vertices, total_new_vertices, sizeof(Vertex), sort_vertices);



        // DIAMETER PRUNING
        number_of_removed_candidates = 0;
        pneighbors_start = graph.twohop_offsets[pvertexid];
        pneighbors_end = graph.twohop_offsets[pvertexid + 1];
        pneighbors_count = pneighbors_end - pneighbors_start;
        for (int j = 1; j < total_new_vertices; j++) {
            phelper1 = new_vertices[j].vertexid;
            phelper2 = binary_search_array(graph.twohop_neighbors + pneighbors_start, pneighbors_count, phelper1);
            if (phelper2 == -1) {
                new_vertices[j].label = -1;
                number_of_removed_candidates++;
            }
        }
        qsort(new_vertices, total_new_vertices, sizeof(Vertex), sort_vertices);

        // update exdeg of vertices connected to removed cands
        for (int i = 0; i < total_new_vertices - number_of_removed_candidates; i++) {
            pvertexid = new_vertices[i].vertexid;
            for (int j = total_new_vertices - number_of_removed_candidates; j < total_new_vertices; j++) {
                phelper1 = new_vertices[j].vertexid;
                pneighbors_start = graph.onehop_offsets[phelper1];
                pneighbors_end = graph.onehop_offsets[phelper1 + 1];
                pneighbors_count = pneighbors_end - pneighbors_start;
                phelper2 = binary_search_array(graph.onehop_neighbors + pneighbors_start, pneighbors_count, pvertexid);
                if (phelper2 != -1) {
                    new_vertices[i].exdeg--;
                }

                pneighbors_start = graph.twohop_offsets[phelper1];
                pneighbors_end = graph.twohop_offsets[phelper1 + 1];
                pneighbors_count = pneighbors_end - pneighbors_start;
                phelper2 = binary_search_array(graph.twohop_neighbors + pneighbors_start, pneighbors_count, pvertexid);
                if (phelper2 != -1) {
                    new_vertices[i].lvl2adj--;
                }
            }
        }
        total_new_vertices -= number_of_removed_candidates;

        // continue if not enough vertices after pruning
        if (total_new_vertices < minimum_clique_size) {
            continue;
        }



        // DEGREE-BASED PRUNING
        do {
            // remove cands that do not meet the deg requirement
            number_of_removed_candidates = 0;
            for (int j = 1; j < total_new_vertices; j++) {
                if (!cand_isvalid(new_vertices[j], 1)) {
                    new_vertices[j].label = -1;
                    number_of_removed_candidates++;
                }
            }
            qsort(new_vertices, total_new_vertices, sizeof(Vertex), sort_vertices);

            // update exdeg of vertices connected to removed cands
            for (int i = 0; i < total_new_vertices - number_of_removed_candidates; i++) {
                pvertexid = new_vertices[i].vertexid;
                for (int j = total_new_vertices - number_of_removed_candidates; j < total_new_vertices; j++) {
                    phelper1 = new_vertices[j].vertexid;
                    pneighbors_start = graph.onehop_offsets[phelper1];
                    pneighbors_end = graph.onehop_offsets[phelper1 + 1];
                    pneighbors_count = pneighbors_end - pneighbors_start;
                    phelper2 = binary_search_array(graph.onehop_neighbors + pneighbors_start, pneighbors_count, pvertexid);
                    if (phelper2 != -1) {
                        new_vertices[i].exdeg--;
                    }

                    pneighbors_start = graph.twohop_offsets[phelper1];
                    pneighbors_end = graph.twohop_offsets[phelper1 + 1];
                    pneighbors_count = pneighbors_end - pneighbors_start;
                    phelper2 = binary_search_array(graph.twohop_neighbors + pneighbors_start, pneighbors_count, pvertexid);
                    if (phelper2 != -1) {
                        new_vertices[i].lvl2adj--;
                    }
                }
            }
            total_new_vertices -= number_of_removed_candidates;
        } while (number_of_removed_candidates > 0);

        // continue if not enough vertices after pruning
        if (total_new_vertices < minimum_clique_size) {
            continue;
        }



        // WRITE TO TASKS
        if (total_new_vertices - 1 > 0) 
        {
            if ((*(host_data.tasks1_count)) < EXPAND_THRESHOLD) {
                uint64_t start_write = host_data.tasks1_offset[(*(host_data.tasks1_count))];

                for (int j = 0; j < total_new_vertices; j++) {
                    host_data.tasks1_vertices[start_write + j].vertexid = new_vertices[j].vertexid;
                    host_data.tasks1_vertices[start_write + j].label = new_vertices[j].label;
                    host_data.tasks1_vertices[start_write + j].indeg = new_vertices[j].indeg;
                    host_data.tasks1_vertices[start_write + j].exdeg = new_vertices[j].exdeg;
                    host_data.tasks1_vertices[start_write + j].lvl2adj = new_vertices[j].lvl2adj;
                }
                (*(host_data.tasks1_count))++;
                host_data.tasks1_offset[(*(host_data.tasks1_count))] = start_write + total_new_vertices;
            }
            else {
                uint64_t start_write = host_data.buffer_offset[(*(host_data.buffer_count))];

                for (int j = 0; j < total_new_vertices; j++) {
                    host_data.buffer_vertices[start_write + j].vertexid = new_vertices[j].vertexid;
                    host_data.buffer_vertices[start_write + j].label = new_vertices[j].label;
                    host_data.buffer_vertices[start_write + j].indeg = new_vertices[j].indeg;
                    host_data.buffer_vertices[start_write + j].exdeg = new_vertices[j].exdeg;
                    host_data.buffer_vertices[start_write + j].lvl2adj = new_vertices[j].lvl2adj;
                }
                (*(host_data.buffer_count))++;
                host_data.buffer_offset[(*(host_data.buffer_count))] = start_write + total_new_vertices;
            }
        }
        delete new_vertices;
    }
    delete old_vertices;
}

void move_to_gpu(CPU_Data& host_data, GPU_Data& dd)
{
    hipMemcpy(dd.tasks1_count, host_data.tasks1_count, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dd.tasks1_offset, host_data.tasks1_offset, (EXPAND_THRESHOLD + 1) * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dd.tasks1_vertices, host_data.tasks1_vertices, (TASKS_SIZE) * sizeof(Vertex), hipMemcpyHostToDevice);

    hipMemcpy(dd.buffer_count, host_data.buffer_count, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dd.buffer_offset, host_data.buffer_offset, (BUFFER_OFFSET_SIZE) * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dd.buffer_vertices, host_data.buffer_vertices, (BUFFER_SIZE) * sizeof(int), hipMemcpyHostToDevice);
}

void dump_cliques(CPU_Cliques& host_cliques, GPU_Data& dd, ofstream& temp_results)
{
    // gpu cliques to cpu cliques
    chkerr(hipMemcpy(host_cliques.cliques_count, dd.cliques_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(host_cliques.cliques_offset, dd.cliques_offset, sizeof(uint64_t) * CLIQUES_OFFSET_SIZE, hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(host_cliques.cliques_vertex, dd.cliques_vertex, sizeof(int) * CLIQUES_SIZE, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    // DEBUG
    //print_CPU_Cliques(host_cliques);

    for (int i = 0; i < ((*host_cliques.cliques_count)); i++) {
        uint64_t start = host_cliques.cliques_offset[i];
        uint64_t end = host_cliques.cliques_offset[i + 1];
        temp_results << end - start << " ";
        for (uint64_t j = start; j < end; j++) {
            temp_results << host_cliques.cliques_vertex[j] << " ";
        }
        temp_results << "\n";
    }
    ((*host_cliques.cliques_count)) = 0;
    hipMemset(dd.cliques_count, 0, sizeof(uint64_t));
}

void free_memory(CPU_Data& host_data, GPU_Data& dd, CPU_Cliques& host_cliques)
{
    // GPU GRAPH
    chkerr(hipFree(dd.number_of_vertices));
    chkerr(hipFree(dd.number_of_edges));
    chkerr(hipFree(dd.onehop_neighbors));
    chkerr(hipFree(dd.onehop_offsets));
    chkerr(hipFree(dd.twohop_neighbors));
    chkerr(hipFree(dd.twohop_offsets));

    // CPU DATA
    delete host_data.tasks1_count;
    delete host_data.tasks1_offset;
    delete host_data.tasks1_vertices;

    delete host_data.buffer_count;
    delete host_data.buffer_offset;
    delete host_data.buffer_vertices;

    delete host_data.maximal_expansion;
    delete host_data.dumping_cliques;

    // GPU DATA
    chkerr(hipFree(dd.current_level));

    chkerr(hipFree(dd.tasks1_count));
    chkerr(hipFree(dd.tasks1_offset));
    chkerr(hipFree(dd.tasks1_vertices));

    chkerr(hipFree(dd.tasks2_count));
    chkerr(hipFree(dd.tasks2_offset));
    chkerr(hipFree(dd.tasks2_vertices));

    chkerr(hipFree(dd.buffer_count));
    chkerr(hipFree(dd.buffer_offset));
    chkerr(hipFree(dd.buffer_vertices));

    chkerr(hipFree(dd.wtasks_count));
    chkerr(hipFree(dd.wtasks_offset));
    chkerr(hipFree(dd.wtasks_vertices));

    chkerr(hipFree(dd.wvertices));

    chkerr(hipFree(dd.maximal_expansion));
    chkerr(hipFree(dd.dumping_cliques));

    chkerr(hipFree(dd.minimum_degree_ratio));
    chkerr(hipFree(dd.minimum_degrees));
    chkerr(hipFree(dd.minimum_clique_size));

    chkerr(hipFree(dd.total_tasks));

    // CPU CLIQUES
    delete host_cliques.cliques_count;
    delete host_cliques.cliques_vertex;
    delete host_cliques.cliques_offset;

    // GPU CLIQUES
    chkerr(hipFree(dd.cliques_count));
    chkerr(hipFree(dd.cliques_vertex));
    chkerr(hipFree(dd.cliques_offset));

    chkerr(hipFree(dd.wcliques_count));
    chkerr(hipFree(dd.wcliques_vertex));
    chkerr(hipFree(dd.wcliques_offset));

    chkerr(hipFree(dd.buffer_offset_start));
    chkerr(hipFree(dd.buffer_start));
    chkerr(hipFree(dd.cliques_offset_start));
    chkerr(hipFree(dd.cliques_start));

    //DEBUG
    chkerr(hipFree(dd.debug));
    chkerr(hipFree(dd.idebug));
}



// --- HELPER METHODS ---

// searches an int array for a certain int, returns the position in the array that item was found, or -1 if not found
int binary_search_array(int* search_array, int array_size, int search_number)
{
    // ALGO - binary
    // TYPE - serial
    // SPEED - 0(log(n))

    if (array_size <= 0) {
        return -1;
    }

    if (search_array[array_size / 2] == search_number) {
        // Base case: Center element matches search number
        return array_size / 2;
    }
    else if (search_array[array_size / 2] > search_number) {
        // Recursively search lower half
        return binary_search_array(search_array, array_size / 2, search_number);
    }
    else {
        // Recursively search upper half
        int upper_half_result = binary_search_array(search_array + array_size / 2 + 1, array_size - array_size / 2 - 1, search_number);
        return (upper_half_result != -1) ? (array_size / 2 + 1 + upper_half_result) : -1;
    }
}

int sort_vertices(const void* a, const void* b)
{
    // order is: in clique -> covered -> critical adj vertices -> cands -> cover -> pruned

    // in clique
    if ((*(Vertex*)a).label == 1 && (*(Vertex*)b).label != 1) {
        return -1;
    }
    else if ((*(Vertex*)a).label != 1 && (*(Vertex*)b).label == 1) {
        return 1;

        // covered candidate vertices
    }
    else if ((*(Vertex*)a).label == 2 && (*(Vertex*)b).label != 2) {
        return -1;
    }
    else if ((*(Vertex*)a).label != 2 && (*(Vertex*)b).label == 2) {
        return 1;

        // critical adjacent candidate vertices
    }
    else if ((*(Vertex*)a).label == 4 && (*(Vertex*)b).label != 4) {
        return -1;
    }
    else if ((*(Vertex*)a).label != 4 && (*(Vertex*)b).label == 4) {
        return 1;

        // candidate vertices
    }
    else if ((*(Vertex*)a).label == 0 && (*(Vertex*)b).label != 0) {
        return -1;
    }
    else if ((*(Vertex*)a).label != 0 && (*(Vertex*)b).label == 0) {
        return 1;

        // the cover vertex
    }
    else if ((*(Vertex*)a).label == 3 && (*(Vertex*)b).label != 3) {
        return -1;
    }
    else if ((*(Vertex*)a).label != 3 && (*(Vertex*)b).label == 3) {
        return 1;

        // vertices that have been pruned
    }
    else if ((*(Vertex*)a).label == -1 && (*(Vertex*)b).label != 1) {
        return 1;
    }
    else if ((*(Vertex*)a).label != -1 && (*(Vertex*)b).label == -1) {
        return -1;
    }

    // for ties: in clique low -> high, cand high -> low
    else if ((*(Vertex*)a).label == 1 && (*(Vertex*)b).label == 1) {
        if ((*(Vertex*)a).vertexid > (*(Vertex*)b).vertexid) {
            return 1;
        }
        else if ((*(Vertex*)a).vertexid < (*(Vertex*)b).vertexid) {
            return -1;
        }
        else {
            return 0;
        }
    }
    else if ((*(Vertex*)a).label == 0 && (*(Vertex*)b).label == 0) {
        if ((*(Vertex*)a).vertexid > (*(Vertex*)b).vertexid) {
            return -1;
        }
        else if ((*(Vertex*)a).vertexid < (*(Vertex*)b).vertexid) {
            return 1;
        }
        else {
            return 0;
        }
    }
    else if ((*(Vertex*)a).label == 2 && (*(Vertex*)b).label == 2) {
        return 0;
    }
    else if ((*(Vertex*)a).label == -1 && (*(Vertex*)b).label == -1) {
        return 0;
    }
    return 0;
}

inline int get_mindeg(int clique_size) {
    if (clique_size < minimum_clique_size) {
        return minimum_degrees[minimum_clique_size];
    }
    else {
        return minimum_degrees[clique_size];
    }
}

inline bool cand_isvalid(Vertex& vertex, int clique_size) {
    if (vertex.indeg + vertex.exdeg < minimum_degrees[minimum_clique_size]) {
        return false;
    }
    else if (vertex.lvl2adj < minimum_clique_size - 1) {
        return false;
    }
    else if (vertex.indeg + vertex.exdeg < get_mindeg(clique_size + vertex.exdeg + 1)) {
        return false;
    }
    else {
        return true;
    }
}

inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        cout << hipGetErrorString(code) << std::endl;
        exit(-1);
    }
}



// --- DEBUG METHODS ---

void print_CPU_Graph(CPU_Graph& host_graph) {
    cout << endl << " --- (CPU_Graph)host_graph details --- " << endl;
    cout << endl << "|V|: " << host_graph.number_of_vertices << " |E|: " << host_graph.number_of_edges << endl;
    cout << endl << "Onehop Offsets:" << endl;
    for (uint64_t i = 0; i <= host_graph.number_of_vertices; i++) {
        cout << host_graph.onehop_offsets[i] << " ";
    }
    cout << endl << "Onehop Neighbors:" << endl;
    for (uint64_t i = 0; i < host_graph.number_of_onehop_neighbors; i++) {
        cout << host_graph.onehop_neighbors[i] << " ";
    }
    cout << endl << "Twohop Offsets:" << endl;
    for (uint64_t i = 0; i <= host_graph.number_of_vertices; i++) {
        cout << host_graph.twohop_offsets[i] << " ";
    }
    cout << endl << "Twohop Neighbors:" << endl;
    for (uint64_t i = 0; i < host_graph.number_of_twohop_neighbors; i++) {
        cout << host_graph.twohop_neighbors[i] << " ";
    }
    cout << endl << endl;
}

void print_GPU_Graph(GPU_Data& dd, CPU_Graph& host_graph)
{
    int* number_of_vertices = new int;
    int* number_of_edges = new int;

    int* onehop_neighbors = new int[host_graph.number_of_onehop_neighbors];
    uint64_t * onehop_offsets = new uint64_t[(host_graph.number_of_vertices)+1];
    int* twohop_neighbors = new int[host_graph.number_of_twohop_neighbors];
    uint64_t * twohop_offsets = new uint64_t[(host_graph.number_of_vertices)+1];

    chkerr(hipMemcpy(number_of_vertices, dd.number_of_vertices, sizeof(int), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(number_of_edges, dd.number_of_edges, sizeof(int), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(onehop_neighbors, dd.onehop_neighbors, sizeof(int)*host_graph.number_of_onehop_neighbors, hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(onehop_offsets, dd.onehop_offsets, sizeof(uint64_t)*(host_graph.number_of_vertices+1), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(twohop_neighbors, dd.twohop_neighbors, sizeof(int)*host_graph.number_of_twohop_neighbors, hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(twohop_offsets, dd.twohop_offsets, sizeof(uint64_t)*(host_graph.number_of_vertices+1), hipMemcpyDeviceToHost));

    cout << endl << " --- (GPU_Graph)device_graph details --- " << endl;
    cout << endl << "|V|: " << (*number_of_vertices) << " |E|: " << (*number_of_edges) << endl;
    cout << endl << "Onehop Offsets:" << endl;
    for (uint64_t i = 0; i <= (*number_of_vertices); i++) {
        cout << onehop_offsets[i] << " ";
    }
    cout << endl << "Onehop Neighbors:" << endl;
    for (uint64_t i = 0; i < host_graph.number_of_onehop_neighbors; i++) {
        cout << onehop_neighbors[i] << " ";
    }
    cout << endl << "Twohop Offsets:" << endl;
    for (uint64_t i = 0; i <= (*number_of_vertices); i++) {
        cout << twohop_offsets[i] << " ";
    }
    cout << endl << "Twohop Neighbors:" << endl;
    for (uint64_t i = 0; i < host_graph.number_of_twohop_neighbors; i++) {
        cout << twohop_neighbors[i] << " ";
    }
    cout << endl << endl;

    delete number_of_vertices;
    delete number_of_edges;

    delete onehop_offsets;
    delete onehop_neighbors;
    delete twohop_offsets;
    delete twohop_neighbors;
}

void print_CPU_Data(CPU_Data& host_data)
{
    cout << endl << " --- (CPU_Data)host_data details --- " << endl;
    cout << endl << "Tasks1: " << "Size: " << (*(host_data.tasks1_count)) << endl;
    cout << endl << "Offsets:" << endl;
    for (uint64_t i = 0; i <= (*(host_data.tasks1_count)); i++) {
        cout << host_data.tasks1_offset[i] << " ";
    }
    cout << endl << "Vertex:" << endl;
    for (uint64_t i = 0; i < host_data.tasks1_offset[(*(host_data.tasks1_count))]; i++) {
        cout << host_data.tasks1_vertices[i].vertexid << " ";
    }
    cout << endl << "Label:" << endl;
    for (uint64_t i = 0; i < host_data.tasks1_offset[(*(host_data.tasks1_count))]; i++) {
        cout << host_data.tasks1_vertices[i].label << " ";
    }
    cout << endl << "Indeg:" << endl;
    for (uint64_t i = 0; i < host_data.tasks1_offset[(*(host_data.tasks1_count))]; i++) {
        cout << host_data.tasks1_vertices[i].indeg << " ";
    }
    cout << endl << "Exdeg:" << endl;
    for (uint64_t i = 0; i < host_data.tasks1_offset[(*(host_data.tasks1_count))]; i++) {
        cout << host_data.tasks1_vertices[i].exdeg << " ";
    }
    cout << endl << "Lvl2adj:" << endl;
    for (uint64_t i = 0; i < host_data.tasks1_offset[(*(host_data.tasks1_count))]; i++) {
        cout << host_data.tasks1_vertices[i].lvl2adj << " ";
    }

    cout << endl << endl << "Buffer: " << "Size: " << (*(host_data.buffer_count)) << endl;
    cout << endl << "Offsets:" << endl;
    for (uint64_t i = 0; i <= (*(host_data.buffer_count)); i++) {
        cout << host_data.buffer_offset[i] << " ";
    }
    cout << endl << "Vertex:" << endl;
    for (uint64_t i = 0; i < host_data.buffer_offset[(*(host_data.buffer_count))]; i++) {
        cout << host_data.buffer_vertices[i].vertexid << " ";
    }
    cout << endl << "Label:" << endl;
    for (uint64_t i = 0; i < host_data.buffer_offset[(*(host_data.buffer_count))]; i++) {
        cout << host_data.buffer_vertices[i].label << " ";
    }
    cout << endl << "Indeg:" << endl;
    for (uint64_t i = 0; i < host_data.buffer_offset[(*(host_data.buffer_count))]; i++) {
        cout << host_data.buffer_vertices[i].indeg << " ";
    }
    cout << endl << "Exdeg:" << endl;
    for (uint64_t i = 0; i < host_data.buffer_offset[(*(host_data.buffer_count))]; i++) {
        cout << host_data.buffer_vertices[i].exdeg << " ";
    }
    cout << endl << "Lvl2adj:" << endl;
    for (uint64_t i = 0; i < host_data.buffer_offset[(*(host_data.buffer_count))]; i++) {
        cout << host_data.buffer_vertices[i].lvl2adj << " ";
    }
    cout << endl << endl;
}

void print_GPU_Data(GPU_Data& dd)
{
    uint64_t* current_level = new uint64_t;

    uint64_t* tasks1_count = new uint64_t;
    uint64_t* tasks1_offset = new uint64_t[EXPAND_THRESHOLD + 1];
    Vertex* tasks1_vertices = new Vertex[TASKS_SIZE];

    uint64_t* tasks2_count = new uint64_t;
    uint64_t* tasks2_offset = new uint64_t[EXPAND_THRESHOLD + 1];
    Vertex* tasks2_vertices = new Vertex[TASKS_SIZE];


    uint64_t* buffer_count = new uint64_t;
    uint64_t* buffer_offset = new uint64_t[BUFFER_OFFSET_SIZE];
    Vertex* buffer_vertices = new Vertex[BUFFER_SIZE];


    chkerr(hipMemcpy(current_level, dd.current_level, sizeof(uint64_t), hipMemcpyDeviceToHost));

    chkerr(hipMemcpy(tasks1_count, dd.tasks1_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(tasks1_offset, dd.tasks1_offset, (EXPAND_THRESHOLD + 1) * sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(tasks1_vertices, dd.tasks1_vertices, (TASKS_SIZE) * sizeof(Vertex), hipMemcpyDeviceToHost));

    chkerr(hipMemcpy(tasks2_count, dd.tasks2_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(tasks2_offset, dd.tasks2_offset, (EXPAND_THRESHOLD + 1) * sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(tasks2_vertices, dd.tasks2_vertices, (TASKS_SIZE) * sizeof(Vertex), hipMemcpyDeviceToHost));

    chkerr(hipMemcpy(buffer_count, dd.buffer_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(buffer_offset, dd.buffer_offset, (BUFFER_OFFSET_SIZE) * sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(buffer_vertices, dd.buffer_vertices, (BUFFER_SIZE) * sizeof(Vertex), hipMemcpyDeviceToHost));

    cout << " --- (GPU_Data)device_data details --- " << endl;
    cout << endl << "Tasks1: Level: " << (*current_level) << " Size: " << (*tasks1_count) << endl;
    cout << endl << "Offsets:" << endl;
    for (int i = 0; i <= (*tasks1_count); i++) {
        cout << tasks1_offset[i] << " " << flush;
    }
    cout << endl << "Vertex:" << endl;
    for (int i = 0; i < tasks1_offset[*tasks1_count]; i++) {
        cout << tasks1_vertices[i].vertexid << " " << flush;
    }
    cout << endl << "Label:" << endl;
    for (int i = 0; i < tasks1_offset[*tasks1_count]; i++) {
        cout << tasks1_vertices[i].label << " " << flush;
    }
    cout << endl << "Indeg:" << endl;
    for (int i = 0; i < tasks1_offset[*tasks1_count]; i++) {
        cout << tasks1_vertices[i].indeg << " " << flush;
    }
    cout << endl << "Exdeg:" << endl;
    for (int i = 0; i < tasks1_offset[*tasks1_count]; i++) {
        cout << tasks1_vertices[i].exdeg << " " << flush;
    }
    cout << endl << "Lvl2adj:" << endl;
    for (int i = 0; i < tasks1_offset[*tasks1_count]; i++) {
        cout << tasks1_vertices[i].lvl2adj << " " << flush;
    }
    cout << endl;

    cout << endl << "Tasks2: " << "Size: " << (*tasks2_count) << endl;
    cout << endl << "Offsets:" << endl;
    for (int i = 0; i <= (*tasks2_count); i++) {
        cout << tasks2_offset[i] << " " << flush;
    }
    cout << endl << "Vertex:" << endl;
    for (int i = 0; i < tasks2_offset[*tasks2_count]; i++) {
        cout << tasks2_vertices[i].vertexid << " " << flush;
    }
    cout << endl << "Label:" << endl;
    for (int i = 0; i < tasks2_offset[*tasks2_count]; i++) {
        cout << tasks2_vertices[i].label << " " << flush;
    }
    cout << endl << "Indeg:" << endl;
    for (int i = 0; i < tasks2_offset[*tasks2_count]; i++) {
        cout << tasks2_vertices[i].indeg << " " << flush;
    }
    cout << endl << "Exdeg:" << endl;
    for (int i = 0; i < tasks2_offset[*tasks2_count]; i++) {
        cout << tasks2_vertices[i].exdeg << " " << flush;
    }
    cout << endl << "Lvl2adj:" << endl;
    for (int i = 0; i < tasks2_offset[*tasks2_count]; i++) {
        cout << tasks2_vertices[i].lvl2adj << " " << flush;
    }
    cout << endl << endl;

    cout << endl << "Buffer: " << "Size: " << (*buffer_count) << endl;
    cout << endl << "Offsets:" << endl;
    for (int i = 0; i <= (*buffer_count); i++) {
        cout << buffer_offset[i] << " " << flush;
    }
    cout << endl << "Vertex:" << endl;
    for (int i = 0; i < buffer_offset[*buffer_count]; i++) {
        cout << buffer_vertices[i].vertexid << " " << flush;
    }
    cout << endl << "Label:" << endl;
    for (int i = 0; i < buffer_offset[*buffer_count]; i++) {
        cout << buffer_vertices[i].label << " " << flush;
    }
    cout << endl << "Indeg:" << endl;
    for (int i = 0; i < buffer_offset[*buffer_count]; i++) {
        cout << buffer_vertices[i].indeg << " " << flush;
    }
    cout << endl << "Exdeg:" << endl;
    for (int i = 0; i < buffer_offset[*buffer_count]; i++) {
        cout << buffer_vertices[i].exdeg << " " << flush;
    }
    cout << endl << "Lvl2adj:" << endl;
    for (int i = 0; i < buffer_offset[*buffer_count]; i++) {
        cout << buffer_vertices[i].lvl2adj << " " << flush;
    }
    cout << endl;

    delete current_level;

    delete tasks1_count;
    delete tasks1_offset;
    delete tasks1_vertices;

    delete tasks2_count;
    delete tasks2_offset;
    delete tasks2_vertices;

    delete buffer_count;
    delete buffer_offset;
    delete buffer_vertices;
}

// CURSOR - test this method, then run program on larger data sets
void print_Warp_Data_Sizes(GPU_Data& dd)
{
    int number_of_warps = (NUM_OF_BLOCKS * BLOCK_SIZE) / WARP_SIZE;

    uint64_t* tasks_counts = new uint64_t[number_of_warps];
    uint64_t* tasks_sizes = new uint64_t[number_of_warps];
    int tasks_tcount = 0;
    int tasks_tsize = 0;
    int tasks_mcount = 0;
    int tasks_msize = 0;
    uint64_t* cliques_counts = new uint64_t[number_of_warps];
    uint64_t* cliques_sizes = new uint64_t[number_of_warps];
    int cliques_tcount = 0;
    int cliques_tsize = 0;
    int cliques_mcount = 0;
    int cliques_msize = 0;

    chkerr(hipMemcpy(tasks_counts, dd.wtasks_count, sizeof(uint64_t) * number_of_warps, hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(cliques_counts, dd.wcliques_count, sizeof(uint64_t) * number_of_warps, hipMemcpyDeviceToHost));
    for (int i = 0; i < number_of_warps; i++) {
        chkerr(hipMemcpy(tasks_sizes + i, dd.wtasks_offset + (i * WTASKS_OFFSET_SIZE) + tasks_counts[i], sizeof(uint64_t), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(cliques_sizes + i, dd.wcliques_offset + (i * WCLIQUES_OFFSET_SIZE) + cliques_counts[i], sizeof(uint64_t), hipMemcpyDeviceToHost));
    }

    for (int i = 0; i < number_of_warps; i++) {
        tasks_tcount += tasks_counts[i];
        if (tasks_counts[i] > tasks_mcount) {
            tasks_mcount = tasks_counts[i];
        }
        tasks_tsize += tasks_sizes[i];
        if (tasks_sizes[i] > tasks_msize) {
            tasks_msize = tasks_sizes[i];
        }
        cliques_tcount += cliques_counts[i];
        if (cliques_counts[i] > cliques_mcount) {
            cliques_mcount = cliques_counts[i];
        }
        cliques_tsize += cliques_sizes[i];
        if (cliques_sizes[i] > cliques_msize) {
            cliques_msize = cliques_sizes[i];
        }
    }

    cout << "WTasks( TC: " << tasks_tcount << " TS: " << tasks_tsize << " MC: " << tasks_mcount << " MS: " << tasks_msize << ") WCliques ( TC: " << cliques_tcount << " TS: " << cliques_tsize << " MC: " << cliques_mcount << " MS: " << cliques_msize << ")" << endl;

    if (tasks_mcount > WTASKS_OFFSET_SIZE || tasks_msize > WTASKS_OFFSET_SIZE || cliques_mcount > WCLIQUES_OFFSET_SIZE || cliques_msize > WCLIQUES_SIZE) {
        cout << "!!! WBUFFER SIZE ERROR !!!" << endl;
    }

    delete tasks_counts;
    delete tasks_sizes;
    delete cliques_counts;
    delete cliques_sizes;
}

// CURSOR - test this method, then run program on larger data sets
void print_All_Warp_Data_Sizes(GPU_Data& dd)
{
    int number_of_warps = (NUM_OF_BLOCKS * BLOCK_SIZE) / WARP_SIZE;

    uint64_t* tasks_counts = new uint64_t[number_of_warps];
    uint64_t* tasks_sizes = new uint64_t[number_of_warps];
    uint64_t* cliques_counts = new uint64_t[number_of_warps];
    uint64_t* cliques_sizes = new uint64_t[number_of_warps];

    chkerr(hipMemcpy(tasks_counts, dd.wtasks_count, sizeof(uint64_t) * number_of_warps, hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(cliques_counts, dd.wcliques_count, sizeof(uint64_t) * number_of_warps, hipMemcpyDeviceToHost));
    for (int i = 0; i < number_of_warps; i++) {
        chkerr(hipMemcpy(tasks_sizes + i, dd.wtasks_offset + (i * WTASKS_OFFSET_SIZE) + tasks_counts[i], sizeof(uint64_t), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(cliques_sizes + i, dd.wcliques_offset + (i * WCLIQUES_OFFSET_SIZE) + cliques_counts[i], sizeof(uint64_t), hipMemcpyDeviceToHost));
    }

    cout << "WTasks Sizes: " << flush;
    for (int i = 0; i < number_of_warps; i++) {
        cout << i << ":" << tasks_counts[i] << " " << tasks_sizes[i] << " " << flush;
    }
    cout << "\nWCliques Sizez: " << flush;
    for (int i = 0; i < number_of_warps; i++) {
        cout << i << ":" << cliques_counts[i] << " " << cliques_sizes[i] << " " << flush;
    }

    delete tasks_counts;
    delete tasks_sizes;
    delete cliques_counts;
    delete cliques_sizes;
}

void print_Warp_Data_Sizes_Every(GPU_Data& dd, int every)
{
    int level;
    chkerr(hipMemcpy(&level, dd.current_level, sizeof(int), hipMemcpyDeviceToHost));
    if (level % every == 0) {
        print_Warp_Data_Sizes(dd);
    }
}

void print_All_Warp_Data_Sizes_Every(GPU_Data& dd, int every)
{
    int level;
    chkerr(hipMemcpy(&level, dd.current_level, sizeof(int), hipMemcpyDeviceToHost));
    if (level % every == 0) {
        print_All_Warp_Data_Sizes(dd);
    }
}

void print_debug(GPU_Data& dd)
{
    bool debug;
    chkerr(hipMemcpy(&debug, dd.debug, sizeof(bool), hipMemcpyDeviceToHost));
    if (debug) {
        cout << "!!!DEBUG!!! " << endl;
    }
    chkerr(hipMemset(dd.debug, false, sizeof(bool)));
}

void print_idebug(GPU_Data& dd)
{
    int idebug;
    chkerr(hipMemcpy(&idebug, dd.idebug, sizeof(int), hipMemcpyDeviceToHost));
    cout << "IDebug: " << idebug << flush;
    chkerr(hipMemset(dd.idebug, 0, sizeof(int)));
}

void print_Data_Sizes_Every(GPU_Data& dd, int every)
{
    int level;
    chkerr(hipMemcpy(&level, dd.current_level, sizeof(int), hipMemcpyDeviceToHost));
    if (level % every == 0) {
        print_Data_Sizes(dd);
    }
}

void print_Data_Sizes(GPU_Data& dd)
{
    uint64_t* current_level = new uint64_t;
    uint64_t* tasks1_count = new uint64_t;
    uint64_t* tasks2_count = new uint64_t;
    uint64_t* buffer_count = new uint64_t;
    uint64_t* cliques_count = new uint64_t;
    uint64_t* tasks1_size = new uint64_t;
    uint64_t* tasks2_size = new uint64_t;
    uint64_t* buffer_size = new uint64_t;
    uint64_t* cliques_size = new uint64_t;

    chkerr(hipMemcpy(current_level, dd.current_level, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(tasks1_count, dd.tasks1_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(tasks2_count, dd.tasks2_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(buffer_count, dd.buffer_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(cliques_count, dd.cliques_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(tasks1_size, dd.tasks1_offset + (*tasks1_count), sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(tasks2_size, dd.tasks2_offset + (*tasks2_count), sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(buffer_size, dd.buffer_offset + (*buffer_count), sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(cliques_size, dd.cliques_offset + (*cliques_count), sizeof(uint64_t), hipMemcpyDeviceToHost));

    cout << "L: " << (*current_level) << " T1: " << (*tasks1_count) << " " << (*tasks1_size) << " T2: " << (*tasks2_count) << " " << (*tasks2_size) << " B: " << (*buffer_count) << " " << (*buffer_size) << " C: " << 
        (*cliques_count) << " " << (*cliques_size) << endl;

    delete current_level;
    delete tasks1_count;
    delete tasks2_count;
    delete buffer_count;
    delete cliques_count;
    delete tasks1_size;
    delete tasks2_size;
    delete buffer_size;
    delete cliques_size;
}

void print_WTask_Buffers(GPU_Data& dd)
{
    int warp_count = (NUM_OF_BLOCKS * BLOCK_SIZE) / 32;
    uint64_t* wtasks_count = new uint64_t[warp_count];
    uint64_t* wtasks_offset = new uint64_t[warp_count*WTASKS_OFFSET_SIZE];
    Vertex* wtasks_vertices = new Vertex[warp_count*WTASKS_SIZE];

    chkerr(hipMemcpy(wtasks_count, dd.wtasks_count, sizeof(uint64_t)*warp_count, hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(wtasks_offset, dd.wtasks_offset, sizeof(uint64_t) * (warp_count*WTASKS_OFFSET_SIZE), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(wtasks_vertices, dd.wtasks_vertices, sizeof(Vertex) * (warp_count*WTASKS_SIZE), hipMemcpyDeviceToHost));

    cout << endl << " --- Warp Task Buffers details --- " << endl;
    for (int i = 0; i < warp_count; i++) {
        int wtasks_offset_start = WTASKS_OFFSET_SIZE * i;
        int wtasks_start = WTASKS_SIZE * i;

        cout << endl << "Warp " << i << ": " << "Size : " << wtasks_count[i] << endl;
        if (wtasks_count[i] == 0) {
            continue;
        }
        cout << "Offsets:" << endl;
        for (int j = 0; j <= wtasks_count[i]; j++) {
            cout << wtasks_offset[wtasks_offset_start+j] << " ";
        }
        cout << endl << "Vertex:" << endl;
        for (int j = 0; j < wtasks_offset[wtasks_offset_start+wtasks_count[i]]; j++) {
            cout << wtasks_vertices[wtasks_start+j].vertexid << " ";
        }
        cout << endl << "Label:" << endl;
        for (int j = 0; j < wtasks_offset[wtasks_offset_start + wtasks_count[i]]; j++) {
            cout << wtasks_vertices[wtasks_start + j].label << " ";
        }
        cout << endl << "Indeg:" << endl;
        for (int j = 0; j < wtasks_offset[wtasks_offset_start + wtasks_count[i]]; j++) {
            cout << wtasks_vertices[wtasks_start + j].indeg << " ";
        }
        cout << endl << "Exdeg:" << endl;
        for (int j = 0; j < wtasks_offset[wtasks_offset_start + wtasks_count[i]]; j++) {
            cout << wtasks_vertices[wtasks_start + j].exdeg << " ";
        }
        cout << endl << "Lvl2adj:" << endl;
        for (int j = 0; j < wtasks_offset[wtasks_offset_start + wtasks_count[i]]; j++) {
            cout << wtasks_vertices[wtasks_start + j].lvl2adj << " ";
        }
        cout << endl;
    }
    cout << endl << endl;

    delete wtasks_count;
    delete wtasks_offset;
    delete wtasks_vertices;
}

void print_WClique_Buffers(GPU_Data& dd)
{
    int warp_count = (NUM_OF_BLOCKS * BLOCK_SIZE) / 32;
    uint64_t* wcliques_count = new uint64_t[warp_count];
    uint64_t* wcliques_offset = new uint64_t[warp_count * WCLIQUES_OFFSET_SIZE];
    int* wcliques_vertex = new int[warp_count * WCLIQUES_SIZE];

    chkerr(hipMemcpy(wcliques_count, dd.wcliques_count, sizeof(uint64_t) * warp_count, hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(wcliques_offset, dd.wcliques_offset, sizeof(uint64_t) * (warp_count * WTASKS_OFFSET_SIZE), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(wcliques_vertex, dd.wcliques_vertex, sizeof(int) * (warp_count * WTASKS_SIZE), hipMemcpyDeviceToHost));

    cout << endl << " --- Warp Clique Buffers details --- " << endl;
    for (int i = 0; i < warp_count; i++) {
        int wcliques_offset_start = WTASKS_OFFSET_SIZE * i;
        int wcliques_start = WTASKS_SIZE * i;

        cout << endl << "Warp " << i << ": " << "Size : " << wcliques_count[i] << endl;
        cout << "Offsets:" << endl;
        for (int j = 0; j <= wcliques_count[i]; j++) {
            cout << wcliques_offset[wcliques_offset_start + j] << " ";
        }
        cout << endl << "Vertex:" << endl;
        for (int j = 0; j < wcliques_offset[wcliques_offset_start + wcliques_count[i]]; j++) {
            cout << wcliques_vertex[wcliques_start + j] << " ";
        }
    }
    cout << endl << endl;

    delete wcliques_count;
    delete wcliques_offset;
    delete wcliques_vertex;
}

void print_GPU_Cliques(GPU_Data& dd)
{
    uint64_t* cliques_count = new uint64_t;
    uint64_t* cliques_offset = new uint64_t[CLIQUES_OFFSET_SIZE];
    int* cliques_vertex = new int[CLIQUES_SIZE];

    chkerr(hipMemcpy(cliques_count, dd.cliques_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(cliques_offset, dd.cliques_offset, sizeof(uint64_t) * CLIQUES_OFFSET_SIZE, hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(cliques_vertex, dd.cliques_vertex, sizeof(int) * CLIQUES_SIZE, hipMemcpyDeviceToHost));

    cout << endl << " --- (GPU_Cliques)device_cliques details --- " << endl;
    cout << endl << "Cliques: " << "Size: " << (*cliques_count) << endl;
    cout << endl << "Offsets:" << endl;
    for (uint64_t i = 0; i <= (*cliques_count); i++) {
        cout << cliques_offset[i] << " ";
    }

    cout << endl << "Vertex:" << endl;
    for (uint64_t i = 0; i < (*cliques_count); i++) {
        cout << i << " S: " << cliques_offset[i] << " E: " << cliques_offset[i+1] << " " << flush;
        for (uint64_t j = cliques_offset[i]; j < cliques_offset[i + 1]; j++) {
            cout << cliques_vertex[j] << " " << flush;
        }
        cout << endl;
    }

    delete cliques_count;
    delete cliques_offset;
    delete cliques_vertex;

    return;

    cout << endl << "Vertex:" << endl;
    for (uint64_t i = 0; i < cliques_offset[(*cliques_count)]; i++) {
        cout << cliques_vertex[i] << " ";
    }
    cout << endl;
}

void print_CPU_Cliques(CPU_Cliques& host_cliques)
{
    cout << endl << " --- (CPU_Cliques)host_cliques details --- " << endl;
    cout << endl << "Cliques: " << "Size: " << (*(host_cliques.cliques_count)) << endl;
    cout << endl << "Offsets:" << endl;
    for (uint64_t i = 0; i <= (*(host_cliques.cliques_count)); i++) {
        cout << host_cliques.cliques_offset[i] << " ";
    }
    cout << endl << "Vertex:" << endl;
    for (uint64_t i = 0; i < host_cliques.cliques_offset[(*(host_cliques.cliques_count))]; i++) {
        cout << host_cliques.cliques_vertex[i] << " ";
    }
    cout << endl;
}

void print_vertices(Vertex* vertices, int size)
{
    cout << " --- level 0 details --- " << endl;
    cout << endl << "Tasks1: Level: " << 0 << " Size: " << size << endl;
    cout << endl << "Offsets:" << endl;
    cout << "0 " << size << flush;
    cout << endl << "Vertex:" << endl;
    for (int i = 0; i < size; i++) {
        cout << vertices[i].vertexid << " " << flush;
    }
    cout << endl << "Label:" << endl;
    for (int i = 0; i < size; i++) {
        cout << vertices[i].label << " " << flush;
    }
    cout << endl << "Indeg:" << endl;
    for (int i = 0; i < size; i++) {
        cout << vertices[i].indeg << " " << flush;
    }
    cout << endl << "Exdeg:" << endl;
    for (int i = 0; i < size; i++) {
        cout << vertices[i].exdeg << " " << flush;
    }
    cout << endl << "Lvl2adj:" << endl;
    for (int i = 0; i < size; i++) {
        cout << vertices[i].lvl2adj << " " << flush;
    }
    cout << endl;
}



// --- DEVICE KERNELS ---

__global__ void expand_level(GPU_Data dd)
{
    // data is stored in data structures to reduce the number of variables that need to be passed to methods
    __shared__ Warp_Data wd;
    Local_Data ld;

    // helper variables, not passed through to any methods
    int method_return;

    // initialize variables
    ld.idx = (blockIdx.x * blockDim.x + threadIdx.x);
    ld.warp_in_block_idx = ((ld.idx / WARP_SIZE) % (BLOCK_SIZE / WARP_SIZE));

    /*
    * The program alternates between reading and writing between to 'tasks' arrays in device global memory. The program will read from one tasks, expand to the next level by generating and pruning, then it will write to the
    * other tasks array. It will write the first EXPAND_THRESHOLD to the tasks array and the rest to the top of the buffer. The buffers acts as a stack containing the excess data not being expanded from tasks. Since the 
    * buffer acts as a stack, in a last-in first-out manner, a subsection of the search space will be expanded until completion. This system allows the problem to essentially be divided into smaller problems and thus 
    * require less memory to handle.
    */
    if ((*(dd.current_level)) % 2 == 1) {
        ld.read_count = dd.tasks1_count;
        ld.read_offsets = dd.tasks1_offset;
        ld.read_vertices = dd.tasks1_vertices;
    } else {
        ld.read_count = dd.tasks2_count;
        ld.read_offsets = dd.tasks2_offset;
        ld.read_vertices = dd.tasks2_vertices;
    }



    // --- CURRENT LEVEL ---
    for (int i = (ld.idx / WARP_SIZE); i < (*(ld.read_count)); i += ((NUM_OF_BLOCKS * BLOCK_SIZE) / WARP_SIZE))
    {
        // get information on vertices being handled within tasks
        if ((ld.idx % WARP_SIZE) == 0) {
            wd.start[ld.warp_in_block_idx] = ld.read_offsets[i];
            wd.end[ld.warp_in_block_idx] = ld.read_offsets[i + 1];
            wd.tot_vert[ld.warp_in_block_idx] = wd.end[ld.warp_in_block_idx] - wd.start[ld.warp_in_block_idx];
            wd.num_mem[ld.warp_in_block_idx] = 0;
            for (uint64_t j = wd.start[ld.warp_in_block_idx]; j < wd.end[ld.warp_in_block_idx]; j++) {
                if (ld.read_vertices[j].label == 1) {
                    wd.num_mem[ld.warp_in_block_idx]++;
                } else {
                    break;
                }
            }
            wd.num_cand[ld.warp_in_block_idx] = wd.tot_vert[ld.warp_in_block_idx] - wd.num_mem[ld.warp_in_block_idx];
            wd.expansions[ld.warp_in_block_idx] = wd.num_cand[ld.warp_in_block_idx];
        }
        __syncwarp();



        // LOOKAHEAD PRUNING
        method_return = lookahead_pruning(dd, wd, ld);
        if (method_return) {
            continue;
        }



        // --- NEXT LEVEL ---
        for (int j = 0; j < wd.expansions[ld.warp_in_block_idx]; j++)
        {


            // REMOVE ONE VERTEX
            if (j > 0) {
                method_return = remove_one_vertex(dd, wd, ld);
                if (method_return) {
                    continue;
                }
            }



            // INITIALIZE NEW VERTICES
            if ((ld.idx % WARP_SIZE) == 0) {
                wd.number_of_members[ld.warp_in_block_idx] = wd.num_mem[ld.warp_in_block_idx];
                wd.number_of_candidates[ld.warp_in_block_idx] = wd.num_cand[ld.warp_in_block_idx];
                wd.total_vertices[ld.warp_in_block_idx] = wd.tot_vert[ld.warp_in_block_idx];
            }
            __syncwarp();

            // select whether to store vertices in global or shared memory based on size
            if (wd.total_vertices[ld.warp_in_block_idx] <= VERTICES_SIZE) {
                ld.vertices = wd.shared_vertices + (VERTICES_SIZE * ld.warp_in_block_idx);
            } else {
                ld.vertices = dd.wvertices + (WVERTICES_SIZE * (ld.idx / WARP_SIZE));
            }

            for (int k = (ld.idx % WARP_SIZE); k < wd.total_vertices[ld.warp_in_block_idx]; k += WARP_SIZE) {
                ld.vertices[k] = ld.read_vertices[wd.start[ld.warp_in_block_idx] + k];
            }



            // ADD ONE VERTEX
            method_return = add_one_vertex(dd, wd, ld);
            // too many vertices pruned continue, no need to check as not enough vertices
            if (method_return == 1) {
                continue;
            }



            // HANDLE CLIQUES
            if (wd.number_of_members[ld.warp_in_block_idx] >= (*dd.minimum_clique_size)) {
                check_for_clique(dd, wd, ld);
            }

            // if vertex in x found as not extendable continue to next iteration
            if (method_return == 2) {
                continue;
            }



            // WRITE TASKS TO BUFFERS
            if (wd.number_of_candidates[ld.warp_in_block_idx] > 0) {
                write_to_tasks(dd, wd, ld);
            }
        }
    }



    if ((ld.idx % WARP_SIZE) == 0) {
        // sum to find tasks count
        atomicAdd(dd.total_tasks, dd.wtasks_count[(ld.idx / WARP_SIZE)]);
        atomicAdd(dd.total_cliques, dd.wcliques_count[(ld.idx / WARP_SIZE)]);
        // DEBUG
        atomicAdd(dd.idebug, dd.wtasks_offset[(WTASKS_OFFSET_SIZE * (ld.idx / WARP_SIZE)) + dd.wtasks_count[(ld.idx / WARP_SIZE)]]);
    }

    if (ld.idx == 0) {
        (*(dd.buffer_offset_start)) = (*(dd.buffer_count)) + 1;
        (*(dd.buffer_start)) = dd.buffer_offset[(*(dd.buffer_count))];
        (*(dd.cliques_offset_start)) = (*(dd.cliques_count)) + 1;
        (*(dd.cliques_start)) = dd.cliques_offset[(*(dd.cliques_count))];
    }
}

__global__ void transfer_buffers(GPU_Data dd)
{
    // THREAD INFO
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_in_block_idx = ((idx / WARP_SIZE) % (BLOCK_SIZE / WARP_SIZE));

    __shared__ uint64_t tasks_write[(BLOCK_SIZE / WARP_SIZE)];
    __shared__ int tasks_offset_write[(BLOCK_SIZE / WARP_SIZE)];
    __shared__ uint64_t cliques_write[(BLOCK_SIZE / WARP_SIZE)];
    __shared__ int cliques_offset_write[(BLOCK_SIZE / WARP_SIZE)];

    __shared__ int twarp;
    __shared__ int toffsetwrite;
    __shared__ int twrite;
    __shared__ int tasks_end;
    
    uint64_t* write_count;
    uint64_t* write_offsets;
    Vertex* write_vertices;

    if ((*(dd.current_level)) % 2 == 1) {
        write_count = dd.tasks2_count;
        write_offsets = dd.tasks2_offset;
        write_vertices = dd.tasks2_vertices;
    }
    else {
        write_count = dd.tasks1_count;
        write_offsets = dd.tasks1_offset;
        write_vertices = dd.tasks1_vertices;
    }

    // block level
    if (threadIdx.x == 0) {
        toffsetwrite = 0;
        twrite = 0;

        for (int i = 0; i < ((NUM_OF_BLOCKS * BLOCK_SIZE) / WARP_SIZE); i++) {
            if (toffsetwrite + dd.wtasks_count[i] >= EXPAND_THRESHOLD) {
                twarp = i;
                break;
            }
            twrite += dd.wtasks_offset[(WTASKS_OFFSET_SIZE * i) + dd.wtasks_count[i]];
            toffsetwrite += dd.wtasks_count[i];
        }
        tasks_end = twrite + dd.wtasks_offset[(WTASKS_OFFSET_SIZE * twarp) +
            (EXPAND_THRESHOLD - toffsetwrite)];
    }
    __syncthreads();

    // warp level
    if ((idx % WARP_SIZE) == 0)
    {
        tasks_write[warp_in_block_idx] = 0;
        tasks_offset_write[warp_in_block_idx] = 1;
        cliques_write[warp_in_block_idx] = 0;
        cliques_offset_write[warp_in_block_idx] = 1;

        for (int i = 0; i < (idx / WARP_SIZE); i++) {
            tasks_offset_write[warp_in_block_idx] += dd.wtasks_count[i];
            tasks_write[warp_in_block_idx] += dd.wtasks_offset[(WTASKS_OFFSET_SIZE * i) + dd.wtasks_count[i]];

            cliques_offset_write[warp_in_block_idx] += dd.wcliques_count[i];
            cliques_write[warp_in_block_idx] += dd.wcliques_offset[(WCLIQUES_OFFSET_SIZE * i) + dd.wcliques_count[i]];
        }
    }
    __syncwarp();

    // TODO - for the next two blocks use two for loops rather than a conditional
    // move to tasks and buffer
    for (int i = (idx % WARP_SIZE) + 1; i <= dd.wtasks_count[(idx / WARP_SIZE)]; i += WARP_SIZE)
    {
        if (tasks_offset_write[warp_in_block_idx] + i - 1 <= EXPAND_THRESHOLD) {
            // to tasks
            write_offsets[tasks_offset_write[warp_in_block_idx] + i - 1] = dd.wtasks_offset[(WTASKS_OFFSET_SIZE * (idx / WARP_SIZE)) + i] + tasks_write[warp_in_block_idx];
        }
        else {
            // to buffer
            dd.buffer_offset[tasks_offset_write[warp_in_block_idx] + i - 2 - EXPAND_THRESHOLD + (*(dd.buffer_offset_start))] = dd.wtasks_offset[(WTASKS_OFFSET_SIZE * (idx / WARP_SIZE)) + i] + 
                tasks_write[warp_in_block_idx] - tasks_end + (*(dd.buffer_start));
        }
    }

    for (int i = (idx % WARP_SIZE); i < dd.wtasks_offset[(WTASKS_OFFSET_SIZE * (idx / WARP_SIZE)) + dd.wtasks_count[(idx / WARP_SIZE)]]; i += WARP_SIZE) {
        if (tasks_write[warp_in_block_idx] + i < tasks_end) {
            // to tasks
            write_vertices[tasks_write[warp_in_block_idx] + i] = dd.wtasks_vertices[(WTASKS_SIZE * (idx / WARP_SIZE)) + i];
        }
        else {
            // to buffer
            dd.buffer_vertices[(*(dd.buffer_start)) + tasks_write[warp_in_block_idx] + i - tasks_end] = dd.wtasks_vertices[(WTASKS_SIZE * (idx / WARP_SIZE)) + i];
        }
    }

    //move to cliques
    for (int i = (idx % WARP_SIZE) + 1; i <= dd.wcliques_count[(idx / WARP_SIZE)]; i += WARP_SIZE) {
        dd.cliques_offset[(*(dd.cliques_offset_start)) + cliques_offset_write[warp_in_block_idx] + i - 2] = dd.wcliques_offset[(WCLIQUES_OFFSET_SIZE * (idx / WARP_SIZE)) + i] + (*(dd.cliques_start)) + 
            cliques_write[warp_in_block_idx];
    }
    for (int i = (idx % WARP_SIZE); i < dd.wcliques_offset[(WCLIQUES_OFFSET_SIZE * (idx / WARP_SIZE)) + dd.wcliques_count[(idx / WARP_SIZE)]]; i += WARP_SIZE) {
        dd.cliques_vertex[(*(dd.cliques_start)) + cliques_write[warp_in_block_idx] + i] = dd.wcliques_vertex[(WCLIQUES_SIZE * (idx / WARP_SIZE)) + i];
    }

    if (idx == 0) {
        // handle tasks and buffer counts
        if ((*dd.total_tasks) <= EXPAND_THRESHOLD) {
            (*write_count) = (*(dd.total_tasks));
        }
        else {
            (*write_count) = EXPAND_THRESHOLD;
            (*(dd.buffer_count)) += ((*(dd.total_tasks)) - EXPAND_THRESHOLD);
        }
        (*(dd.cliques_count)) += (*(dd.total_cliques));

        (*(dd.total_tasks)) = 0;
        (*(dd.total_cliques)) = 0;
    }

    // HANDLE CLIQUES
    // only first thread for each warp
    if ((idx % WARP_SIZE) == 0 && cliques_write[warp_in_block_idx] > (CLIQUES_SIZE * (CLIQUES_PERCENT / 100.0))) {
        atomicExch((int*)dd.dumping_cliques, true);
    }
}

__global__ void fill_from_buffer(GPU_Data dd)
{
    // THREAD INFO
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_idx = (idx / 32);
    int lane_idx = (idx % 32);

    Vertex* write_vertices;
    uint64_t* write_offsets;
    uint64_t* write_count;

    if ((*(dd.current_level)) % 2 == 1) {
        write_count = dd.tasks2_count;
        write_offsets = dd.tasks2_offset;
        write_vertices = dd.tasks2_vertices;
    } else {
        write_count = dd.tasks1_count;
        write_offsets = dd.tasks1_offset;
        write_vertices = dd.tasks1_vertices;
    }

    if (lane_idx == 0) {
        dd.wtasks_count[warp_idx] = 0;
        dd.wcliques_count[warp_idx] = 0;
    }

    // FILL TASKS FROM BUFFER
    if ((*write_count) < EXPAND_THRESHOLD && (*(dd.buffer_count)) > 0)
    {
        // CRITICAL
        atomicExch((int*)dd.maximal_expansion, false);

        // get read and write locations
        int write_amount = ((*(dd.buffer_count)) >= (EXPAND_THRESHOLD - (*write_count))) ? EXPAND_THRESHOLD - (*write_count) : (*(dd.buffer_count));
        uint64_t start_buffer = dd.buffer_offset[(*(dd.buffer_count)) - write_amount];
        uint64_t end_buffer = dd.buffer_offset[(*(dd.buffer_count))];
        uint64_t size_buffer = end_buffer - start_buffer;
        uint64_t start_write = write_offsets[(*write_count)];

        // handle offsets
        for (int i = idx + 1; i <= write_amount; i += (NUM_OF_BLOCKS * BLOCK_SIZE)) {
            write_offsets[(*write_count) + i] = start_write + (dd.buffer_offset[(*(dd.buffer_count)) - write_amount + i] - start_buffer);
        }

        // handle data
        for (int i = idx; i < size_buffer; i += (NUM_OF_BLOCKS * BLOCK_SIZE)) {
            write_vertices[start_write + i] = dd.buffer_vertices[start_buffer + i];
        }

        if (idx == 0) {
            (*write_count) += write_amount;
            (*(dd.buffer_count)) -= write_amount;
        }
    }

    if (idx == 0) {
        (*dd.current_level)++;
    }
}

// returns 1 if lookahead succesful, 0 otherwise
__device__ int lookahead_pruning(GPU_Data& dd, Warp_Data& wd, Local_Data& ld) 
{
    bool lookahead_sucess = true;

    // compares all vertices to the lemmas from Quick
    for (int j = (ld.idx % WARP_SIZE); j < wd.tot_vert[ld.warp_in_block_idx]; j += WARP_SIZE) {
        if (ld.read_vertices[wd.start[ld.warp_in_block_idx] + j].lvl2adj != (wd.tot_vert[ld.warp_in_block_idx] - 1) || ld.read_vertices[wd.start[ld.warp_in_block_idx] + j].indeg + 
            ld.read_vertices[wd.start[ld.warp_in_block_idx] + j].exdeg < dd.minimum_degrees[wd.tot_vert[ld.warp_in_block_idx]]) {
            lookahead_sucess = false;
            break;
        }
    }
    lookahead_sucess = !(__any_sync(0xFFFFFFFF, !lookahead_sucess));

    if (lookahead_sucess) {
        // write to cliques
        uint64_t start_write = (WCLIQUES_SIZE * (ld.idx / WARP_SIZE)) + dd.wcliques_offset[(WCLIQUES_OFFSET_SIZE * (ld.idx / WARP_SIZE)) + (dd.wcliques_count[(ld.idx / WARP_SIZE)])];
        for (int j = (ld.idx % WARP_SIZE); j < wd.tot_vert[ld.warp_in_block_idx]; j += WARP_SIZE) {
            dd.wcliques_vertex[start_write + j] = ld.read_vertices[wd.start[ld.warp_in_block_idx] + j].vertexid;
        }
        if ((ld.idx % WARP_SIZE) == 0) {
            (dd.wcliques_count[(ld.idx / WARP_SIZE)])++;
            dd.wcliques_offset[(WCLIQUES_OFFSET_SIZE * (ld.idx / WARP_SIZE)) + (dd.wcliques_count[(ld.idx / WARP_SIZE)])] = start_write - (WCLIQUES_SIZE * (ld.idx / WARP_SIZE)) + wd.tot_vert[ld.warp_in_block_idx];
        }
        return 1;
    }
    return 0;
}

// returns 1 if failed found after removing, 0 otherwise
__device__ int remove_one_vertex(GPU_Data& dd, Warp_Data& wd, Local_Data& ld) 
{
    int pvertexid;
    bool failed_found;

    // remove the last candidate in vertices
    if ((ld.idx % WARP_SIZE) == 0) {
        wd.num_cand[ld.warp_in_block_idx]--;
        wd.tot_vert[ld.warp_in_block_idx]--;
    }
    __syncwarp();

    // get the id of the removed vertex and update the degrees of its adjacencies
    pvertexid = ld.read_vertices[wd.start[ld.warp_in_block_idx] + wd.tot_vert[ld.warp_in_block_idx]].vertexid;
    for (int k = (ld.idx % WARP_SIZE); k < wd.tot_vert[ld.warp_in_block_idx]; k += WARP_SIZE) {
        if (device_bsearch_array(dd.onehop_neighbors + dd.onehop_offsets[pvertexid], dd.onehop_offsets[pvertexid + 1] - dd.onehop_offsets[pvertexid], ld.read_vertices[wd.start[ld.warp_in_block_idx] + k].vertexid) != -1) {
            ld.read_vertices[wd.start[ld.warp_in_block_idx] + k].exdeg--;
        }

        if (device_bsearch_array(dd.twohop_neighbors + dd.twohop_offsets[pvertexid], dd.twohop_offsets[pvertexid + 1] - dd.twohop_offsets[pvertexid], ld.read_vertices[wd.start[ld.warp_in_block_idx] + k].vertexid) != -1) {
            ld.read_vertices[wd.start[ld.warp_in_block_idx] + k].lvl2adj--;
        }
    }
    __syncwarp();

    // check for failed vertices
    failed_found = false;
    for (int k = (ld.idx % WARP_SIZE); k < wd.num_mem[ld.warp_in_block_idx]; k += WARP_SIZE) {
        if (!device_vert_isextendable(ld.read_vertices[wd.start[ld.warp_in_block_idx] + k], wd.num_mem[ld.warp_in_block_idx], dd)) {
            failed_found = true;
            break;
        }

    }
    failed_found = __any_sync(0xFFFFFFFF, failed_found);
    if (failed_found) {
        return 1;
    }
    return 0;
}

// returns 2, if too many vertices pruned to be considered, 1 if failed found or invalid bound, 0 otherwise
__device__ int add_one_vertex(GPU_Data& dd, Warp_Data& wd, Local_Data& ld) 
{
    int pvertexid;
    bool failed_found;

    if ((ld.idx % WARP_SIZE) == 0) {
        ld.vertices[wd.total_vertices[ld.warp_in_block_idx] - 1].label = 1;
        wd.number_of_members[ld.warp_in_block_idx]++;
        wd.number_of_candidates[ld.warp_in_block_idx]--;
    }
    __syncwarp();

    // update the exdeg and indeg of all vertices adj to the vertex just added to the vertex set
    pvertexid = ld.vertices[wd.total_vertices[ld.warp_in_block_idx] - 1].vertexid;
    for (int k = (ld.idx % WARP_SIZE); k < wd.total_vertices[ld.warp_in_block_idx]; k += WARP_SIZE) {
        if (device_bsearch_array(dd.onehop_neighbors + dd.onehop_offsets[ld.vertices[k].vertexid], dd.onehop_offsets[ld.vertices[k].vertexid + 1] - dd.onehop_offsets[ld.vertices[k].vertexid], pvertexid) != -1) {
            ld.vertices[k].exdeg--;
            ld.vertices[k].indeg++;
        }
    }
    __syncwarp();

    // TODO - this might be able to be hard coded rather than sorted
    // sort new vertices putting just added vertex at end of all vertices in x
    device_sort(ld.vertices + wd.number_of_members[ld.warp_in_block_idx] - 1, wd.number_of_candidates[ld.warp_in_block_idx] + 1, (ld.idx % WARP_SIZE));



    // --- DIAMETER PRUNING ---
    diameter_pruning(dd, wd, ld, pvertexid);

    // continue if not enough vertices after pruning
    if (wd.total_vertices[ld.warp_in_block_idx] < (*(dd.minimum_clique_size))) {
        return 1;
    }



    // DEGREE BASED PRUNING
    degree_pruning(dd, wd, ld, failed_found);

    // continue if not enough vertices after pruning
    if (wd.total_vertices[ld.warp_in_block_idx] < (*(dd.minimum_clique_size))) {
        return 1;
    }

    // TODO - test if we need to check vertex sets that have invalid bounds, dont think so
    // if vertex in x found as not extendable continue to next iteration
    if (failed_found || wd.invalid_bounds[ld.warp_in_block_idx]) {
        return 2;
    }
    
    return 0;
}

__device__ void check_for_clique(GPU_Data& dd, Warp_Data& wd, Local_Data& ld)
{
    bool clique = true;

    for (int k = (ld.idx % WARP_SIZE); k < wd.number_of_members[ld.warp_in_block_idx]; k += WARP_SIZE) {
        if (ld.vertices[k].indeg < dd.minimum_degrees[wd.number_of_members[ld.warp_in_block_idx]]) {
            clique = false;
            break;
        }
    }
    // set to false if any threads in warp do not meet degree requirement
    clique = !(__any_sync(0xFFFFFFFF, !clique));

    // if clique write to warp buffer for cliques
    if (clique) {
        uint64_t start_write = (WCLIQUES_SIZE * (ld.idx / WARP_SIZE)) + dd.wcliques_offset[(WCLIQUES_OFFSET_SIZE * (ld.idx / WARP_SIZE)) + (dd.wcliques_count[(ld.idx / WARP_SIZE)])];
        for (int k = (ld.idx % WARP_SIZE); k < wd.number_of_members[ld.warp_in_block_idx]; k += WARP_SIZE) {
            dd.wcliques_vertex[start_write + k] = ld.vertices[k].vertexid;
        }
        if ((ld.idx % WARP_SIZE) == 0) {
            (dd.wcliques_count[(ld.idx / WARP_SIZE)])++;
            dd.wcliques_offset[(WCLIQUES_OFFSET_SIZE * (ld.idx / WARP_SIZE)) + (dd.wcliques_count[(ld.idx / WARP_SIZE)])] = start_write - (WCLIQUES_SIZE * (ld.idx / WARP_SIZE)) +
                wd.number_of_members[ld.warp_in_block_idx];
        }
    }
}

__device__ void write_to_tasks(GPU_Data& dd, Warp_Data& wd, Local_Data& ld)
{
    // CRITICAL
    atomicExch((int*)dd.maximal_expansion, false);

    uint64_t start_write = (WTASKS_SIZE * (ld.idx / WARP_SIZE)) + dd.wtasks_offset[WTASKS_OFFSET_SIZE * (ld.idx / WARP_SIZE) + (dd.wtasks_count[(ld.idx / WARP_SIZE)])];

    for (int k = (ld.idx % WARP_SIZE); k < wd.total_vertices[ld.warp_in_block_idx]; k += WARP_SIZE) {
        dd.wtasks_vertices[start_write + k].vertexid = ld.vertices[k].vertexid;
        dd.wtasks_vertices[start_write + k].label = ld.vertices[k].label;
        dd.wtasks_vertices[start_write + k].indeg = ld.vertices[k].indeg;
        dd.wtasks_vertices[start_write + k].exdeg = ld.vertices[k].exdeg;
        dd.wtasks_vertices[start_write + k].lvl2adj = ld.vertices[k].lvl2adj;
    }
    if ((ld.idx % WARP_SIZE) == 0) {
        (dd.wtasks_count[(ld.idx / WARP_SIZE)])++;
        dd.wtasks_offset[(WTASKS_OFFSET_SIZE * (ld.idx / WARP_SIZE)) + (dd.wtasks_count[(ld.idx / WARP_SIZE)])] = start_write - (WTASKS_SIZE * (ld.idx / WARP_SIZE)) + wd.total_vertices[ld.warp_in_block_idx];
    }
}

__device__ void diameter_pruning(GPU_Data& dd, Warp_Data& wd, Local_Data& ld, int pvertexid)
{
    int number_of_removed_candidates;

    number_of_removed_candidates = 0;
    for (int k = wd.number_of_members[ld.warp_in_block_idx] + (ld.idx % WARP_SIZE); k < wd.total_vertices[ld.warp_in_block_idx]; k += WARP_SIZE) {
        if (device_bsearch_array(dd.twohop_neighbors + dd.twohop_offsets[pvertexid], dd.twohop_offsets[pvertexid + 1] - dd.twohop_offsets[pvertexid], ld.vertices[k].vertexid) == -1) {
            ld.vertices[k].label = -1;
            number_of_removed_candidates++;
        }
    }
    for (int k = 1; k < 32; k *= 2) {
        number_of_removed_candidates += __shfl_xor_sync(0xFFFFFFFF, number_of_removed_candidates, k);
    }
    device_sort(ld.vertices + wd.number_of_members[ld.warp_in_block_idx], wd.number_of_candidates[ld.warp_in_block_idx], (ld.idx % WARP_SIZE));

    // update exdeg of vertices connected to removed cands
    update_degrees(dd, wd, ld, number_of_removed_candidates);

    if ((ld.idx % WARP_SIZE) == 0) {
        wd.total_vertices[ld.warp_in_block_idx] -= number_of_removed_candidates;
        wd.number_of_candidates[ld.warp_in_block_idx] -= number_of_removed_candidates;
    }
    __syncwarp();
}

// TODO - return failed_found rather than use a reference
__device__ void degree_pruning(GPU_Data& dd, Warp_Data& wd, Local_Data& ld, bool& failed_found)
{
    int number_of_removed_candidates;

    do
    {
        // calculate lower and upper bounds for vertices
        calculate_LU_bounds(dd, wd, ld);

        if (wd.invalid_bounds[ld.warp_in_block_idx]) {
            break;
        }

        // check for failed vertices
        failed_found = false;
        for (int k = (ld.idx % WARP_SIZE); k < wd.number_of_members[ld.warp_in_block_idx]; k += WARP_SIZE) {
            if (!device_vert_isextendable_LU(ld.vertices[k], dd, wd, ld)) {
                failed_found = true;
                break;
            }

        }
        failed_found = __any_sync(0xFFFFFFFF, failed_found);
        if (failed_found) {
            break;
        }

        // remove cands that do not meet the deg requirement
        number_of_removed_candidates = 0;
        for (int k = wd.number_of_members[ld.warp_in_block_idx] + (ld.idx % WARP_SIZE); k < wd.total_vertices[ld.warp_in_block_idx]; k += WARP_SIZE) {
            if (!device_cand_isvalid_LU(ld.vertices[k], dd, wd, ld)) {
                ld.vertices[k].label = -1;
                number_of_removed_candidates++;
            }
        }
        for (int k = 1; k < 32; k *= 2) {
            number_of_removed_candidates += __shfl_xor_sync(0xFFFFFFFF, number_of_removed_candidates, k);
        }
        device_sort(ld.vertices + wd.number_of_members[ld.warp_in_block_idx], wd.number_of_candidates[ld.warp_in_block_idx], (ld.idx % WARP_SIZE));

        // update exdeg of vertices connected to removed cands
        update_degrees(dd, wd, ld, number_of_removed_candidates);

        if ((ld.idx % WARP_SIZE) == 0) {
            wd.total_vertices[ld.warp_in_block_idx] -= number_of_removed_candidates;
            wd.number_of_candidates[ld.warp_in_block_idx] -= number_of_removed_candidates;
        }
        __syncwarp();
    } while (number_of_removed_candidates > 0);
}

__device__ void update_degrees(GPU_Data& dd, Warp_Data& wd, Local_Data& ld, int number_of_removed_candidates)
{
    int pvertexid;

    /*
    * Program updates degrees by: for each vertex, for each removed vertex, binary search neighbors of removed vertex for (non-removed)vertex. This is an improvement from the Quick algorithm because it uses binary search.
    * Additonally the program also dyanmically selects which for loop to parallelize based on which one is larger, this is the pupose of the if statement.
    */
    if (wd.total_vertices[ld.warp_in_block_idx] - number_of_removed_candidates > number_of_removed_candidates) {
        for (int k = (ld.idx % WARP_SIZE); k < wd.total_vertices[ld.warp_in_block_idx] - number_of_removed_candidates; k += WARP_SIZE) {
            pvertexid = ld.vertices[k].vertexid;
            for (int l = wd.total_vertices[ld.warp_in_block_idx] - number_of_removed_candidates; l < wd.total_vertices[ld.warp_in_block_idx]; l++) {
                if (device_bsearch_array(dd.onehop_neighbors + dd.onehop_offsets[ld.vertices[l].vertexid], dd.onehop_offsets[ld.vertices[l].vertexid + 1] - dd.onehop_offsets[ld.vertices[l].vertexid], pvertexid) != -1) {
                    ld.vertices[k].exdeg--;
                }

                if (device_bsearch_array(dd.twohop_neighbors + dd.twohop_offsets[ld.vertices[l].vertexid], dd.twohop_offsets[ld.vertices[l].vertexid + 1] - dd.twohop_offsets[ld.vertices[l].vertexid], pvertexid) != -1) {
                    ld.vertices[k].lvl2adj--;
                }
            }
        }
        __syncwarp();
    }
    else {
        for (int k = 0; k < wd.total_vertices[ld.warp_in_block_idx] - number_of_removed_candidates; k++) {
            pvertexid = ld.vertices[k].vertexid;
            for (int l = wd.total_vertices[ld.warp_in_block_idx] - number_of_removed_candidates + (ld.idx % WARP_SIZE); l < wd.total_vertices[ld.warp_in_block_idx]; l += WARP_SIZE) {
                if (device_bsearch_array(dd.onehop_neighbors + dd.onehop_offsets[ld.vertices[l].vertexid], dd.onehop_offsets[ld.vertices[l].vertexid + 1] - dd.onehop_offsets[ld.vertices[l].vertexid], pvertexid) != -1) {
                    ld.vertices[k].exdeg--;
                }

                if (device_bsearch_array(dd.twohop_neighbors + dd.twohop_offsets[ld.vertices[l].vertexid], dd.twohop_offsets[ld.vertices[l].vertexid + 1] - dd.twohop_offsets[ld.vertices[l].vertexid], pvertexid) != -1) {
                    ld.vertices[k].lvl2adj--;
                }
            }
            __syncwarp();
        }
    }
}

// TODO - try to parallelize as much calculation as possible
__device__ void calculate_LU_bounds(GPU_Data& dd, Warp_Data& wd, Local_Data& ld)
{
    int index;

    int min_clq_indeg;
    int min_indeg_exdeg;
    int min_clq_totaldeg;
    int sum_clq_indeg;

    // initialize the values of the LU calculation variables to the first vertices values so they can be compared to other vertices without error
    min_clq_indeg = ld.vertices[0].indeg;
    min_indeg_exdeg = ld.vertices[0].exdeg;
    min_clq_totaldeg = ld.vertices[0].indeg + ld.vertices[0].exdeg;
    sum_clq_indeg = 0;

    // each warp also has a copy of these variables to allow for intra-warp comparison of these variables.
    if ((ld.idx % WARP_SIZE) == 0) {
        wd.invalid_bounds[ld.warp_in_block_idx] = false;

        wd.sum_candidate_indeg[ld.warp_in_block_idx] = 0;
        wd.tightened_Upper_bound[ld.warp_in_block_idx] = 0;

        wd.min_clq_indeg[ld.warp_in_block_idx] = ld.vertices[0].indeg;
        wd.min_indeg_exdeg[ld.warp_in_block_idx] = ld.vertices[0].exdeg;
        wd.min_clq_totaldeg[ld.warp_in_block_idx] = ld.vertices[0].indeg + ld.vertices[0].exdeg;
        wd.sum_clq_indeg[ld.warp_in_block_idx] = ld.vertices[0].indeg;

        wd.minimum_external_degree[ld.warp_in_block_idx] = device_get_mindeg(wd.number_of_members[ld.warp_in_block_idx] + 1,
            dd);
    }
    __syncwarp();

    // each warp finds these values on their subsection of vertices
    for (index = 1 + (ld.idx % WARP_SIZE); index < wd.number_of_members[ld.warp_in_block_idx]; index += WARP_SIZE) {
        sum_clq_indeg += ld.vertices[index].indeg;

        if (ld.vertices[index].indeg < min_clq_indeg) {
            min_clq_indeg = ld.vertices[index].indeg;
            min_indeg_exdeg = ld.vertices[index].exdeg;
        }
        else if (ld.vertices[index].indeg == min_clq_indeg) {
            if (ld.vertices[index].exdeg < min_indeg_exdeg) {
                min_indeg_exdeg = ld.vertices[index].exdeg;
            }
        }

        if (ld.vertices[index].indeg + ld.vertices[index].exdeg < min_clq_totaldeg) {
            min_clq_totaldeg = ld.vertices[index].indeg + ld.vertices[index].exdeg;
        }
    }

    // get sum
    for (int i = 1; i < 32; i *= 2) {
        sum_clq_indeg += __shfl_xor_sync(0xFFFFFFFF, sum_clq_indeg, i);
    }
    if ((ld.idx % WARP_SIZE) == 0) {
        // add to shared memory sum
        wd.sum_clq_indeg[ld.warp_in_block_idx] += sum_clq_indeg;
    }
    __syncwarp();

    // CRITICAL SECTION - each lane then compares their values to the next to get a warp level value
    for (int i = 0; i < WARP_SIZE; i++) {
        if ((ld.idx % WARP_SIZE) == i) {
            if (min_clq_indeg < wd.min_clq_indeg[ld.warp_in_block_idx]) {
                wd.min_clq_indeg[ld.warp_in_block_idx] = min_clq_indeg;
                wd.min_indeg_exdeg[ld.warp_in_block_idx] = min_indeg_exdeg;
            }
            else if (min_clq_indeg == wd.min_clq_indeg[ld.warp_in_block_idx]) {
                if (min_indeg_exdeg < wd.min_indeg_exdeg[ld.warp_in_block_idx]) {
                    wd.min_indeg_exdeg[ld.warp_in_block_idx] = min_indeg_exdeg;
                }
            }

            if (min_clq_totaldeg < wd.min_clq_totaldeg[ld.warp_in_block_idx]) {
                wd.min_clq_totaldeg[ld.warp_in_block_idx] = min_clq_totaldeg;
            }
        }
        __syncwarp();
    }

    // TODO - CRITICAL SECTION - unsure how to parallelize this, very complex, determine whether this section is worth having at all
    if ((ld.idx % WARP_SIZE) == 0) {
        if (wd.min_clq_indeg[ld.warp_in_block_idx] < dd.minimum_degrees[wd.number_of_members[ld.warp_in_block_idx]])
        {
            // lower
            wd.Lower_bound[ld.warp_in_block_idx] = device_get_mindeg(wd.number_of_members[ld.warp_in_block_idx], dd) - min_clq_indeg;

            while (wd.Lower_bound[ld.warp_in_block_idx] <= wd.min_indeg_exdeg[ld.warp_in_block_idx] && wd.min_clq_indeg[ld.warp_in_block_idx] + wd.Lower_bound[ld.warp_in_block_idx] <
                dd.minimum_degrees[wd.number_of_members[ld.warp_in_block_idx] + wd.Lower_bound[ld.warp_in_block_idx]]) {
                wd.Lower_bound[ld.warp_in_block_idx]++;
            }

            if (wd.min_clq_indeg[ld.warp_in_block_idx] + wd.Lower_bound[ld.warp_in_block_idx] < dd.minimum_degrees[wd.number_of_members[ld.warp_in_block_idx] + wd.Lower_bound[ld.warp_in_block_idx]]) {
                wd.invalid_bounds[ld.warp_in_block_idx] = true;
            }

            // upper
            wd.Upper_bound[ld.warp_in_block_idx] = floor(wd.min_clq_totaldeg[ld.warp_in_block_idx] / (*(dd.minimum_degree_ratio))) + 1 - wd.number_of_members[ld.warp_in_block_idx];

            if (wd.Upper_bound[ld.warp_in_block_idx] > wd.number_of_candidates[ld.warp_in_block_idx]) {
                wd.Upper_bound[ld.warp_in_block_idx] = wd.number_of_candidates[ld.warp_in_block_idx];
            }

            // tighten
            if (wd.Lower_bound[ld.warp_in_block_idx] < wd.Upper_bound[ld.warp_in_block_idx]) {
                // tighten lower
                for (index = 0; index < wd.Lower_bound[ld.warp_in_block_idx]; index++) {
                    wd.sum_candidate_indeg[ld.warp_in_block_idx] += ld.vertices[wd.number_of_members[ld.warp_in_block_idx] + index].indeg;
                }

                while (index < wd.Upper_bound[ld.warp_in_block_idx] && wd.sum_clq_indeg[ld.warp_in_block_idx] + wd.sum_candidate_indeg[ld.warp_in_block_idx] < wd.number_of_members[ld.warp_in_block_idx] *
                    dd.minimum_degrees[wd.number_of_members[ld.warp_in_block_idx] + index]) {
                    wd.sum_candidate_indeg[ld.warp_in_block_idx] += ld.vertices[wd.number_of_members[ld.warp_in_block_idx] + index].indeg;
                    index++;
                }

                if (wd.sum_clq_indeg[ld.warp_in_block_idx] + wd.sum_candidate_indeg[ld.warp_in_block_idx] < wd.number_of_members[ld.warp_in_block_idx] * dd.minimum_degrees[wd.number_of_members[ld.warp_in_block_idx] + index]) {
                    wd.invalid_bounds[ld.warp_in_block_idx] = true;
                }
                else {
                    wd.Lower_bound[ld.warp_in_block_idx] = index;

                    wd.tightened_Upper_bound[ld.warp_in_block_idx] = index;

                    while (index < wd.Upper_bound[ld.warp_in_block_idx]) {
                        wd.sum_candidate_indeg[ld.warp_in_block_idx] += ld.vertices[wd.number_of_members[ld.warp_in_block_idx] + index].indeg;

                        index++;

                        if (wd.sum_clq_indeg[ld.warp_in_block_idx] + wd.sum_candidate_indeg[ld.warp_in_block_idx] >= wd.number_of_members[ld.warp_in_block_idx] * 
                            dd.minimum_degrees[wd.number_of_members[ld.warp_in_block_idx] + index]) {
                            wd.tightened_Upper_bound[ld.warp_in_block_idx] = index;
                        }
                    }

                    if (wd.Upper_bound[ld.warp_in_block_idx] > wd.tightened_Upper_bound[ld.warp_in_block_idx]) {
                        wd.Upper_bound[ld.warp_in_block_idx] = wd.tightened_Upper_bound[ld.warp_in_block_idx];
                    }

                    if (wd.Lower_bound[ld.warp_in_block_idx] > 1) {
                        wd.minimum_external_degree[ld.warp_in_block_idx] = device_get_mindeg(wd.number_of_members[ld.warp_in_block_idx] + wd.Lower_bound[ld.warp_in_block_idx], dd);
                    }
                }
            }
        }
        else {
            wd.minimum_external_degree[ld.warp_in_block_idx] = device_get_mindeg(wd.number_of_members[ld.warp_in_block_idx] + 1,
                dd);

            wd.Upper_bound[ld.warp_in_block_idx] = wd.number_of_candidates[ld.warp_in_block_idx];

            if (wd.number_of_members[ld.warp_in_block_idx] < (*(dd.minimum_clique_size))) {
                wd.Lower_bound[ld.warp_in_block_idx] = (*(dd.minimum_clique_size)) - wd.number_of_members[ld.warp_in_block_idx];
            }
            else {
                wd.Lower_bound[ld.warp_in_block_idx] = 0;
            }
        }

        if (wd.number_of_members[ld.warp_in_block_idx] + wd.Upper_bound[ld.warp_in_block_idx] < (*(dd.minimum_clique_size))) {
            wd.invalid_bounds[ld.warp_in_block_idx] = true;
        }

        if (wd.Upper_bound[ld.warp_in_block_idx] < 0 || wd.Upper_bound[ld.warp_in_block_idx] < wd.Lower_bound[ld.warp_in_block_idx]) {
            wd.invalid_bounds[ld.warp_in_block_idx] = true;
        }
    }
    __syncwarp();
}



// --- HELPER KERNELS ---

// DEBUG
__device__ void degree_pruning_nonLU(GPU_Data& dd, Warp_Data& wd, Local_Data& ld, bool& failed_found)
{
    int number_of_removed_candidates;

    do
    {
        // check for failed vertices
        failed_found = false;
        for (int k = (ld.idx % WARP_SIZE); k < wd.number_of_members[ld.warp_in_block_idx]; k += WARP_SIZE) {
            if (!device_vert_isextendable(ld.vertices[k], wd.number_of_members[ld.warp_in_block_idx], dd)) {
                failed_found = true;
                break;
            }

        }
        failed_found = __any_sync(0xFFFFFFFF, failed_found);
        if (failed_found) {
            break;
        }

        // remove cands that do not meet the deg requirement
        number_of_removed_candidates = 0;
        for (int k = wd.number_of_members[ld.warp_in_block_idx] + (ld.idx % WARP_SIZE); k < wd.total_vertices[ld.warp_in_block_idx]; k += WARP_SIZE) {
            if (!device_cand_isvalid(ld.vertices[k], wd.number_of_members[ld.warp_in_block_idx], dd)) {
                ld.vertices[k].label = -1;
                number_of_removed_candidates++;
            }
        }
        for (int k = 1; k < 32; k *= 2) {
            number_of_removed_candidates += __shfl_xor_sync(0xFFFFFFFF, number_of_removed_candidates, k);
        }
        device_sort(ld.vertices + wd.number_of_members[ld.warp_in_block_idx], wd.number_of_candidates[ld.warp_in_block_idx], (ld.idx % WARP_SIZE));

        // update exdeg of vertices connected to removed cands
        update_degrees(dd, wd, ld, number_of_removed_candidates);

        if ((ld.idx % WARP_SIZE) == 0) {
            wd.total_vertices[ld.warp_in_block_idx] -= number_of_removed_candidates;
            wd.number_of_candidates[ld.warp_in_block_idx] -= number_of_removed_candidates;
        }
        __syncwarp();
    } while (number_of_removed_candidates > 0);
}

// TODO - convert to merge or radix sort, merge is recursive
__device__ void device_sort(Vertex* target, int size, int lane_idx)
{
    // ALGO - ODD/EVEN
    // TYPE - PARALLEL
    // SPEED - O(n^2)

    for (int i = 0; i < size; i++) {
        for (int j = (i % 2) + (lane_idx * 2); j < size - 1; j += (WARP_SIZE * 2)) {
            Vertex vertex1 = target[j];
            Vertex vertex2 = target[j + 1];

            if (sort_vert(vertex1, vertex2) == 1) {
                target[j] = target[j + 1];
                target[j + 1] = vertex1;
            }
        }
        __syncwarp();
    }
}

// TODO - clean up method
__device__ __forceinline int sort_vert(Vertex& vertex1, Vertex& vertex2)
{
    // order is: in clique -> covered -> critical adj vertices -> cands -> cover -> pruned

    // in clique
    if (vertex1.label == 1 && vertex2.label != 1) {
        return -1;
    }
    else if (vertex1.label != 1 && vertex2.label == 1) {
        return 1;

    // covered candidate vertices
    }
    else if (vertex1.label == 2 && vertex2.label != 2) {
        return -1;
    }
    else if (vertex1.label != 2 && vertex2.label == 2) {
        return 1;

    // critical adjacent candidate vertices
    }
    else if (vertex1.label == 4 && vertex2.label != 4) {
        return -1;
    }
    else if (vertex1.label != 4 && vertex2.label == 4) {
        return 1;

    // candidate vertices
    }
    else if (vertex1.label == 0 && vertex2.label != 0) {
        return -1;
    }
    else if (vertex1.label != 0 && vertex2.label == 0) {
        return 1;

    // the cover vertex
    }
    else if (vertex1.label == 3 && vertex2.label != 3) {
        return -1;
    }
    else if (vertex1.label != 3 && vertex2.label == 3) {
        return 1;

    // vertices that have been pruned
    }
    else if (vertex1.label == -1 && vertex2.label != 1) {
        return 1;
    }
    else if (vertex1.label != -1 && vertex2.label == -1) {
        return -1;
    }

    // for ties: in clique low -> high, cand high -> low
    else if (vertex1.label == 1 && vertex2.label == 1) {
        if (vertex1.vertexid > vertex2.vertexid) {
            return 1;
        }
        else if (vertex1.vertexid < vertex2.vertexid) {
            return -1;
        }
        else {
            return 0;
        }
    }
    else if (vertex1.label == 0 && vertex2.label == 0) {
        if (vertex1.vertexid > vertex2.vertexid) {
            return -1;
        }
        else if (vertex1.vertexid < vertex2.vertexid) {
            return 1;
        }
        else {
            return 0;
        }
    }
    else if (vertex1.label == 2 && vertex2.label == 2) {
        return 0;
    }
    else if (vertex1.label == -1 && vertex2.label == -1) {
        return 0;
    }
    return 0;
}

// searches an int array for a certain int, returns the position in the array that item was found, or -1 if not found
__device__ int device_bsearch_array(int* search_array, int array_size, int search_number)
{
    // ALGO - binary
    // TYPE - serial
    // SPEED - 0(log(n))

    if (array_size <= 0) {
        return -1;
    }

    if (search_array[array_size / 2] == search_number) {
        // Base case: Center element matches search number
        return array_size / 2;
    }
    else if (search_array[array_size / 2] > search_number) {
        // Recursively search lower half
        return device_bsearch_array(search_array, array_size / 2, search_number);
    }
    else {
        // Recursively search upper half
        int upper_half_result = device_bsearch_array(search_array + array_size / 2 + 1, array_size - array_size / 2 - 1, search_number);
        return (upper_half_result != -1) ? (array_size / 2 + 1 + upper_half_result) : -1;
    }
}

__device__ __forceinline bool device_vert_isextendable(Vertex& vertex, int number_of_members, GPU_Data& dd)
{
    if (vertex.indeg + vertex.exdeg < dd.minimum_degrees[(*(dd.minimum_clique_size))]) {
        return false;
    }
    else if (vertex.lvl2adj < (*(dd.minimum_clique_size)) - 1) {
        return false;
    }
    else if (vertex.indeg + vertex.exdeg < device_get_mindeg(number_of_members+vertex.exdeg, dd)) {
        return false;
    }
    else {
        return true;
    }
}

__device__ __forceinline bool device_cand_isvalid(Vertex& vertex, int number_of_members, GPU_Data& dd)
{
    if (vertex.indeg + vertex.exdeg < dd.minimum_degrees[(*(dd.minimum_clique_size))]) {
        return false;
    }
    else if (vertex.lvl2adj < (*(dd.minimum_clique_size)) - 1) {
        return false;
    }
    else if (vertex.indeg + vertex.exdeg < device_get_mindeg(number_of_members + vertex.exdeg + 1, dd)) {
        return false;
    }
    else {
        return true;
    }
}

__device__ __forceinline bool device_cand_isvalid_LU(Vertex& vertex, GPU_Data& dd, Warp_Data& wd, Local_Data& ld)
{
    if (vertex.indeg + vertex.exdeg < dd.minimum_degrees[(*(dd.minimum_clique_size))]) {
        return false;
    }
    else if (vertex.lvl2adj < (*(dd.minimum_clique_size)) - 1) {
        return false;
    }
    else if (vertex.indeg + vertex.exdeg < device_get_mindeg(wd.number_of_members[ld.warp_in_block_idx] + vertex.exdeg + 1, dd)) {
        return false;
    }
    else if (vertex.indeg + vertex.exdeg < wd.minimum_external_degree[ld.warp_in_block_idx]) {
        return false;
    }
    else if (vertex.indeg + wd.Upper_bound[ld.warp_in_block_idx] - 1 < dd.minimum_degrees[wd.number_of_members[ld.warp_in_block_idx] + wd.Lower_bound[ld.warp_in_block_idx]]) {
        return false;
    }
    else if (vertex.indeg + vertex.exdeg < device_get_mindeg(wd.number_of_members[ld.warp_in_block_idx] + wd.Lower_bound[ld.warp_in_block_idx], dd)) {
        return false;
    }
    else {
        return true;
    }
}

__device__ __forceinline bool device_vert_isextendable_LU(Vertex& vertex, GPU_Data& dd, Warp_Data& wd, Local_Data& ld)
{
    if (vertex.indeg + vertex.exdeg < dd.minimum_degrees[(*(dd.minimum_clique_size))]) {
        return false;
    }
    else if (vertex.lvl2adj < (*(dd.minimum_clique_size)) - 1) {
        return false;
    }
    else if (vertex.indeg + vertex.exdeg < device_get_mindeg(wd.number_of_members[ld.warp_in_block_idx] + vertex.exdeg, dd)) {
        return false;
    }
    else if (vertex.indeg + vertex.exdeg < wd.minimum_external_degree[ld.warp_in_block_idx]) {
        return false;
    }
    // TODO - I think this else if is useless
    else if (vertex.exdeg == 0 && vertex.indeg < device_get_mindeg(wd.number_of_members[ld.warp_in_block_idx] + vertex.exdeg, dd)) {
        return false;
    }
    else if (vertex.indeg + wd.Upper_bound[ld.warp_in_block_idx] < dd.minimum_degrees[wd.number_of_members[ld.warp_in_block_idx] + wd.Upper_bound[ld.warp_in_block_idx]]) {
        return false;
    }
    else if (vertex.indeg + vertex.exdeg < device_get_mindeg(wd.number_of_members[ld.warp_in_block_idx] + wd.Lower_bound[ld.warp_in_block_idx], dd)) {
        return false;
    }
    else {
        return true;
    }
}

__device__ __forceinline int device_get_mindeg(int number_of_members, GPU_Data& dd)
{
    if (number_of_members < (*(dd.minimum_clique_size))) {
        return dd.minimum_degrees[(*(dd.minimum_clique_size))];
    }
    else {
        return dd.minimum_degrees[number_of_members];
    }
}



// --- RM NON-MAX (from Quick) ---

int comp_int(const void* e1, const void* e2)
{
    int n1, n2;
    n1 = *(int*)e1;
    n2 = *(int*)e2;

    if (n1 > n2)
        return 1;
    else if (n1 < n2)
        return -1;
    else
        return 0;
}

extern int gntotal_max_cliques;

struct TREE_NODE
{
    int nid;
    TREE_NODE* pchild;
    TREE_NODE* pright_sib;
    bool bis_max;
};

#define TNODE_PAGE_SIZE (1<<10)

struct TNODE_PAGE
{
    TREE_NODE ptree_nodes[TNODE_PAGE_SIZE];
    TNODE_PAGE* pnext;
};

struct TNODE_BUF
{
    TNODE_PAGE* phead;
    TNODE_PAGE* pcur_page;
    int ncur_pos;
    int ntotal_pages;
};

extern TNODE_BUF gotreenode_buf;

inline TREE_NODE* NewTreeNode()
{
    TREE_NODE* ptnode;
    TNODE_PAGE* pnew_page;

    if (gotreenode_buf.ncur_pos == TNODE_PAGE_SIZE)
    {
        if (gotreenode_buf.pcur_page->pnext == NULL)
        {
            pnew_page = new TNODE_PAGE;
            pnew_page->pnext = NULL;
            gotreenode_buf.pcur_page->pnext = pnew_page;
            gotreenode_buf.pcur_page = pnew_page;
            gotreenode_buf.ntotal_pages++;
        }
        else
            gotreenode_buf.pcur_page = gotreenode_buf.pcur_page->pnext;
        gotreenode_buf.ncur_pos = 0;
    }

    ptnode = &(gotreenode_buf.pcur_page->ptree_nodes[gotreenode_buf.ncur_pos]);
    gotreenode_buf.ncur_pos++;

    ptnode->bis_max = true;

    return ptnode;
}

inline void OutputOneSet(FILE* fp, int* pset, int nlen)
{
    int i;

    gntotal_max_cliques++;

    fprintf(fp, "%d ", nlen);
    for (i = 0; i < nlen; i++)
        fprintf(fp, "%d ", pset[i]);
    fprintf(fp, "\n");

}

#include <stdio.h>
#include <time.h>
#include <sys/timeb.h>

int gntotal_max_cliques;

TNODE_BUF gotreenode_buf;

void DelTNodeBuf()
{
    TNODE_PAGE* ppage;

    ppage = gotreenode_buf.phead;
    while (ppage != NULL)
    {
        gotreenode_buf.phead = gotreenode_buf.phead->pnext;
        delete ppage;
        gotreenode_buf.ntotal_pages--;
        ppage = gotreenode_buf.phead;
    }
    if (gotreenode_buf.ntotal_pages != 0)
        printf("Error: inconsistent number of pages\n");
}

void InsertOneSet(int* pset, int nlen, TREE_NODE*& proot)
{
    TREE_NODE* pnode, * pparent, * pleftsib, * pnew_node;
    int i, j;

    i = 0;
    pparent = NULL;
    pnode = proot;
    pleftsib = NULL;

    while (i < nlen)
    {
        while (pnode != NULL && pnode->nid < pset[i])
        {
            pleftsib = pnode;
            pnode = pnode->pright_sib;
        }

        if (pnode == NULL || pnode->nid > pset[i])
        {
            pnew_node = NewTreeNode();
            pnew_node->nid = pset[i];
            pnew_node->pchild = NULL;
            pnew_node->pright_sib = pnode;
            if (pleftsib != NULL)
                pleftsib->pright_sib = pnew_node;
            else if (pparent != NULL)
                pparent->pchild = pnew_node;
            if (i == 0 && pleftsib == NULL)
                proot = pnew_node;
            pparent = pnew_node;
            for (j = i + 1; j < nlen; j++)
            {
                pnew_node = NewTreeNode();
                pnew_node->nid = pset[j];
                pnew_node->pchild = NULL;
                pnew_node->pright_sib = NULL;
                pparent->pchild = pnew_node;
                pparent = pnew_node;
            }
            break;
        }
        else
        {
            pparent = pnode;
            pnode = pnode->pchild;
            pleftsib = NULL;
        }
        i++;
    }
}

int BuildTree(char* szset_filename, TREE_NODE*& proot)
{
    FILE* fp;
    int nlen, * pset, nset_size, i, nmax_len, num_of_sets;

    fp = fopen(szset_filename, "rt");
    if (fp == NULL)
    {
        printf("Error: cannot open file %s for read\n", szset_filename);
        return 0;
    }

    gotreenode_buf.phead = new TNODE_PAGE;
    gotreenode_buf.phead->pnext = NULL;
    gotreenode_buf.pcur_page = gotreenode_buf.phead;
    gotreenode_buf.ntotal_pages = 1;
    gotreenode_buf.ncur_pos = 0;

    proot = NULL;

    num_of_sets = 0;

    nset_size = 100;
    pset = new int[nset_size];

    nmax_len = 0;
    fscanf(fp, "%d", &nlen);
    while (!feof(fp))
    {
        if (nmax_len < nlen)
            nmax_len = nlen;
        if (nlen > nset_size)
        {
            delete[]pset;
            nset_size *= 2;
            if (nset_size < nlen)
                nset_size = nlen;
            pset = new int[nset_size];
        }
        for (i = 0; i < nlen; i++)
            fscanf(fp, "%d", &pset[i]);
        qsort(pset, nlen, sizeof(int), comp_int);
        InsertOneSet(pset, nlen, proot);

        num_of_sets++;
        fscanf(fp, "%d", &nlen);
    }
    fclose(fp);

    delete[]pset;

    return nmax_len;
}

void SearchSubset(int* pset, int nset_len, TREE_NODE* proot, TREE_NODE** pstack, int* ppos)
{
    TREE_NODE* pnode;
    int ntop, npos;

    if (proot == NULL)
        return;
    ntop = 0;
    npos = 0;
    pnode = proot;

    while (ntop >= 0)
    {
        while (pnode != NULL && npos < nset_len && pnode->nid != pset[npos])
        {
            if (pnode->nid < pset[npos])
                pnode = pnode->pright_sib;
            else
                npos++;
        }
        if (pnode != NULL && npos < nset_len)
        {
            if (pnode->pchild == NULL && pnode->bis_max)
                pnode->bis_max = false;
            pstack[ntop] = pnode;
            ppos[ntop] = npos;
            ntop++;
            pnode = pnode->pchild;
            npos++;
        }
        else
        {
            ntop--;
            if (ntop >= 0)
            {
                pnode = pstack[ntop]->pright_sib;
                npos = ppos[ntop] + 1;
            }
        }
    }

}

void RmNonMax(TREE_NODE* proot, int nmax_len)
{
    TREE_NODE* pnode, ** pstack, ** psearch_stack;
    int* pset, ntop, i, * ppos;

    pset = new int[nmax_len];
    pstack = new TREE_NODE * [nmax_len];
    psearch_stack = new TREE_NODE * [nmax_len];
    ppos = new int[nmax_len];

    pstack[0] = proot;
    pset[0] = proot->nid;
    ntop = 1;
    pnode = proot;

    while (ntop > 0)
    {
        if (pnode->pchild != NULL)
        {
            pnode = pnode->pchild;
            pstack[ntop] = pnode;
            pset[ntop] = pnode->nid;
            ntop++;
        }
        else
        {
            if (ntop >= 2 && pnode->bis_max)
            {
                for (i = ntop - 1; i >= 1; i--)
                {
                    if (pstack[i - 1]->pright_sib != NULL)
                        SearchSubset(&pset[i], ntop - i, pstack[i - 1]->pright_sib, psearch_stack, ppos);
                }
            }

            while (ntop > 0 && pnode->pright_sib == NULL)
            {
                ntop--;
                if (ntop > 0)
                    pnode = pstack[ntop - 1];
            }
            if (ntop == 0)
                break;
            else //if(pnode->pright_sib!=NULL)
            {
                pnode = pnode->pright_sib;
                pstack[ntop - 1] = pnode;
                pset[ntop - 1] = pnode->nid;
            }
        }
    }

    delete[]pset;
    delete[]pstack;
    delete[]psearch_stack;
    delete[]ppos;
}

void OutputMaxSet(TREE_NODE* proot, int nmax_len, char* szoutput_filename)
{
    FILE* fp;
    TREE_NODE** pstack, * pnode;
    int* pset, ntop;

    fp = fopen(szoutput_filename, "wt");
    if (fp == NULL)
    {
        printf("Error: cannot open file %s for write\n", szoutput_filename);
        return;
    }

    pstack = new TREE_NODE * [nmax_len];
    pset = new int[nmax_len];

    pstack[0] = proot;
    pset[0] = proot->nid;
    ntop = 1;
    pnode = proot;

    while (ntop > 0)
    {
        if (pnode->pchild != NULL)
        {
            pnode = pnode->pchild;
            pstack[ntop] = pnode;
            pset[ntop] = pnode->nid;
            ntop++;
        }
        else
        {
            if (pnode->bis_max)
                OutputOneSet(fp, pset, ntop);

            while (ntop > 0 && pnode->pright_sib == NULL)
            {
                ntop--;
                if (ntop > 0)
                    pnode = pstack[ntop - 1];
            }
            if (ntop == 0)
                break;
            else //if(pnode->pright_sib!=NULL)
            {
                pnode = pnode->pright_sib;
                pstack[ntop - 1] = pnode;
                pset[ntop - 1] = pnode->nid;
            }
        }
    }

    delete[]pstack;
    delete[]pset;

    fclose(fp);
}

void RemoveNonMax(char* szset_filename, char* szoutput_filename)
{
    cout << ">:REMOVING NON-MAXIMAL CLIQUES" << endl;

    TREE_NODE* proot;
    int nmax_len;
    struct timeb start, end;

    ftime(&start);

    gntotal_max_cliques = 0;

    nmax_len = BuildTree(szset_filename, proot);
    RmNonMax(proot, nmax_len);
    OutputMaxSet(proot, nmax_len, szoutput_filename);

    DelTNodeBuf();

    ftime(&end);


    printf(">:NUMBER OF MAXIMAL CLIQUES: %d\n", gntotal_max_cliques);
}