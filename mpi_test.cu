#include "./inc/device_funcs.h"
#include "./inc/gpu_memory_allocation.h"
#include "mpi.h"

#define PATHLEN 10
#define MAXNODES 8
#define MAXGB 1000000000

struct Point {
    unsigned int a, b, c, d, e;
};

/*s
TYPES OF MESSAGES AND THEIR MEANING:
c - taker thread asking for confirmation from giver thread
r - giver thread is requesting another thread to help take some of its work
C - giver thread giving confirmation to taker that it will send work
t - taker thread letting others know it has found worksss
f - taker thread letting others know it is free and could recieve work
D - giver thrread declining to confirm to a taker that it will send work
* z - not an actual message but used as a place holder to indicate that
*/

int wsize;
int grank;
// for every task there is a seperate message buffer and incoming/outgoing handle slot
char msg_buffer[MAXNODES][100];
// array of handles for messages with all other thread, allows for asynchronous messaging, handles say whether message is complete
MPI_Request rq_send_msg[MAXNODES];
MPI_Request rq_recv_msg[MAXNODES];
unsigned int iter = 0;
bool global_free_list[MAXNODES];

// checks for error when running CUDA method
inline void chkerr(hipError_t code) {
    if (code != hipSuccess) {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<std::endl;
        exit(-1);
    }
}

// counts how many threads are done working in the program, if all threads are done can end the program
int count_free_list() {
    int cnt = 0;
    for (int i = 0; i < wsize; ++i) {
        if (global_free_list[i]) {
            cnt++;
        }
    }
    return cnt;
}

// asynchronously recieve a 1 char message from src thread in msg_buffer with a handle rq_recv_msg
void mpi_irecv(int src) {
    MPI_Irecv(msg_buffer[src], 1, MPI_CHAR, src, 0, MPI_COMM_WORLD,
              &rq_recv_msg[src]);

}

// asynchronously send a 1 char message to dest thread in msg_buffer with a handle rq_send_msg
void mpi_isend(int dest, char *msg) {
    //MPI_Isend(msg, strlen(msg) + 1, MPI_CHAR, dest, 0, MPI_COMM_WORLD,
    MPI_Isend(msg, 1, MPI_CHAR, dest, 0, MPI_COMM_WORLD,
              &rq_send_msg[dest]);

}

// asynchronously recieves messages from all other threads in world
void mpi_irecv_all(int rank) {
    for (int i = 0; i < wsize; i++) {
        if (i != rank) {
            mpi_irecv(i);
        }
    }
}

// asynchronously sends messages to all other threads in world
void mpi_isend_all(int rank, char *msg) {
    for (int i = 0; i < wsize; i++) {
        if (i != rank) {
            mpi_isend(i, msg);
        }
    }
}

// attempts to recieve work from another thread, returns true if work recieved else false
bool take_work(int from, int rank, unsigned int *buffer) {
    /// first ask the other node to confirm that it has pending work
    /// it might have finished it by the time we received the processing request or
    /// someone else might have offered it help

    // asks for confirmation form thread "from"
    mpi_isend(from, "c");

    MPI_Status status;

    // initialize last message to default value
    char last_msg = 'r';

    // keep getting messages from "from" thread until a recieved message is no longer a request message
    while (last_msg == 'r') // the while loop ensures that multiple `r' requests are removed
    {
        MPI_Wait(&rq_recv_msg[from], &status); //blocking wait till we get a proper response
        last_msg = msg_buffer[from][0];
        mpi_irecv(from); ///initiate a request
    }

    // when repsponce is recieved check whether from has work or not
    // if there is work on from
    if (last_msg == 'C') {

        // let other threads know this thread found work
        mpi_isend_all(rank, "t");

        // initialize a new MPI dataype of 5 unsigned ints called dt_point
        MPI_Datatype dt_point;
        MPI_Type_contiguous(5, MPI_UNSIGNED, &dt_point);
        MPI_Type_commit(&dt_point);

        // recieve MAXGB dt_points from the "from" thread, these dt_points will be interpreted as Point*
        // NOTE - this seems to be their data format, our equivalent would probably be Vertex
        MPI_Recv((Point *) buffer, MAXGB, dt_point, from, 1, MPI_COMM_WORLD, &status);
        
        // set current rank in free list as false
        global_free_list[rank] = false;

        return true;

        // if there is no more work on from
    } else if (last_msg == 'f') {

        // set from thread in free list as true
        global_free_list[from] = true;
    }

    return false;
}

// sends message to all other threads indicating it can recieve work, when it find a giver thread it requests confirmation
int take_work_wrap(int rank, unsigned int *buffer) {

    bool took_work = false;

    // send message to all threads that current thread is free
    mpi_isend_all(rank, "f");

    // set index in free list as true
    global_free_list[rank] = true;

    // until current thread has taken work from another or all threads are done
    while (!took_work && count_free_list() < wsize)
    {

        // for all other threads
        for (int i = 0; i < wsize; i++) {
            if (i == rank) {
                continue;
            }

            // does nothing, just needed for test call
            MPI_Status status;

            // initialize flag and last message to default values
            int flag = 1;
            char last_msg = 'z'; //invalid message

            // while we have still recieved a message from another thread i
            while (flag == 1)
            {
                // get the next message from thread i
                MPI_Test(&rq_recv_msg[i], &flag, &status); //check if we recvd a msg

                // if there was a message
                if (flag) {

                    // get the message
                    last_msg = msg_buffer[i][0];

                    // try to get he next message
                    mpi_irecv(i);

                    // if the message was that thread i was empty mark it as such
                    if (last_msg == 'f') {
                        global_free_list[i] = true;

                    // uf the message was that thread i got new work mark it as such
                    } else if (last_msg == 't') {
                        global_free_list[i] = false;
                    }
                }
            }
            if (last_msg == 'r')//someone is asking us to help to process their request...
            {

                // have to check whether we've taken work because, when we do we will still check messages form other threads
                // this is to see if they are also reporting free
                if (!took_work) {
                    took_work = take_work(i, rank, buffer);
                }
            }
        }
    }

    // return how many threads are done
    return count_free_list();
}

// actually sends work from current thread to "taker" thread
void give_work(int rank, int taker, unsigned int *buffer) {

    // not used, just needed as parameter
    MPI_Status status;

    // send C as indicating confirmation to taker thread
    mpi_isend(taker, "C");

    // wait until C is fully sent
    MPI_Wait(&rq_send_msg[taker], &status);
    /// At this point we know that the taker is waiting to recv data
    /// TODO WRITE CODE HERE to initiate data transfer
    /// USE TAG 1 for sync

    unsigned int giveSize;

    // UNSURE - how much is this, will be important for our program as well
    giveSize = ((buffer[0] + buffer[1])*2 + 2) / 5 + 1;

    // declare new MPI data type
    MPI_Datatype dt_point;
    MPI_Type_contiguous(5, MPI_UNSIGNED, &dt_point);
    MPI_Type_commit(&dt_point);

    // send data
    MPI_Send((Point *) buffer, giveSize, dt_point, taker, 1, MPI_COMM_WORLD);
}

// looks to see if another thread is requesting confirmation for transfer, if so transfers data to it and declines other threads askign for data
// the taker parameter is really a return value of the thread id of the thread we gave work to
bool check_for_confirmation(int rank, int &taker, unsigned int *buffer) {

    bool agreed_to_split_work = false;

    /// first try to respond all nodes which has send a confirmation request as all of them will be waiting
    // iterate through all other threads
    for (int i = 0; i < wsize; i++) {
        if (i == rank) {
            continue;
        }

        // not used, needed as parameter
        MPI_Status status;

        // initialize loop parameters
        int flag = true;
        char last_msg = 'z'; //invalid message

        // while there are still messages from thread i
        while (flag) /// move forward till we find the last message
        {

            // check if the current thread has recieved a message from thread i
            MPI_Test(&rq_recv_msg[i], &flag, &status); //check if we recvd a msg

            // if we recieved a message
            if (flag) {

                // get the last message from thread i
                last_msg = msg_buffer[i][0];

                // if the last message is f then mark thread i as free in the free list
                if (last_msg == 'f') {
                    global_free_list[i] = true;

                // if the last message is that the thread has taken work mark thread i as not free in the free list
                } else if (last_msg == 't') {
                    global_free_list[i] = false;
                }

                // recieve the next message
                mpi_irecv(i); /// initiate new recv request again
            }
        }

        // if the last message from some taker thread was asking for for data
        if (last_msg == 'c') //we found someone waiting for confirmation
        {

            // and we haven't given work to another thread yet
            if (!agreed_to_split_work) {

                // give work to the taker thread
                give_work(rank, i, buffer); //give work to this node
                agreed_to_split_work = true;

                // set the return variable taker as the id of thread we gave the work to
                taker = i;

            // we have already given work to another thread
            } else {

                ///send decline
                // send a declination message to the taker thread asking for data
                mpi_isend(i, "D");
            }
        }
    }

    // return whether we were able to give work to someone else
    // NOTE - I don't think we will need this with how our adaptation might work
    return agreed_to_split_work;
}

// check to see if a previous request for help was responded to, then send  another request for help and see if anyone repsonds
bool give_work_wrapper(int rank, int &taker, unsigned int *buffer) {

    // see if any thread has asked for confirmation from a previously sent request, if so this method also sends the data
    bool agreed_to_split_work = check_for_confirmation(rank, taker, buffer);


    /// no one send confirmation
    // if no one has sent a confirmation previously
    if (!agreed_to_split_work) {

        // for all other threads if they are currently free send a request for help
        for (int i = 0; i < wsize; i++) /// send a process request to all free nodes
        {
            if (i != rank && global_free_list[i]) {

                // the message for requesting for help
                mpi_isend(i, "r");
            }
        }

        /// retry to see someone send confirmation
        // now that new messages have been sent see if any thread is asking for confirmation
        agreed_to_split_work = check_for_confirmation(rank, taker, buffer);
    }

    // return whether work was split with another thread
    return agreed_to_split_work;
}

// UNSURE - not sure how their data encoding works, don't think it is important as we will send our data differently either way
// seems like the first three elements are size data and the rest is the actual data
void encode_com_buffer(unsigned int *mpi_buffer,S_pointers s,unsigned iter,unsigned int buf_len){
    unsigned int pre_len = s.lengths[iter - 1];
    mpi_buffer[0] = pre_len;
    mpi_buffer[1] = buf_len;
    mpi_buffer[2] = iter;
    unsigned int copy_offset = 3;
    chkerr(hipMemcpy(&mpi_buffer[copy_offset], s.results_table,pre_len * sizeof(unsigned int),
                      hipMemcpyDeviceToHost));
    copy_offset+=(pre_len);
    chkerr(hipMemcpy(&mpi_buffer[copy_offset], &s.results_table[pre_len+buf_len],
                      buf_len * sizeof(unsigned int),hipMemcpyDeviceToHost));
    copy_offset+=buf_len;
    chkerr(hipMemcpy(&mpi_buffer[copy_offset],s.indexes_table,pre_len * sizeof(unsigned int),
                      hipMemcpyDeviceToHost));
    copy_offset+=pre_len;
    chkerr(hipMemcpy(&mpi_buffer[copy_offset],&s.indexes_table[pre_len+buf_len],
                      buf_len * sizeof(unsigned int),hipMemcpyDeviceToHost));
}

// UNSURE - not sure how their data encoding works, don't think it is important as we will send our data differently either way
// seems like the first three elements are size data and the rest is the actual data
unsigned int decode_com_buffer(unsigned int *mpi_buffer,S_pointers &s){
    unsigned int pre_len = mpi_buffer[0];
    unsigned int buf_len = mpi_buffer[1];
    unsigned int iter = mpi_buffer[2];
    unsigned int copy_offset = 3;
    chkerr(hipMemcpy(s.results_table,&mpi_buffer[copy_offset],(pre_len+buf_len) * sizeof(unsigned int),
                      hipMemcpyHostToDevice));
    copy_offset+=(pre_len+buf_len);
    chkerr(hipMemcpy(s.indexes_table,&mpi_buffer[copy_offset],(pre_len+buf_len) * sizeof(unsigned int),
                      hipMemcpyHostToDevice));
    s.lengths[iter - 1] = pre_len;
    s.lengths[iter] = s.lengths[iter - 1] + buf_len;
    return iter;
}

// method is unimportant just decided whether to launch kernels with virutal warps or normal warps
void kernel_launch(G_pointers query_pointers,G_pointers data_pointers,C_pointers c_pointers,S_pointers &s_pointers,
                   unsigned int U,unsigned int iter,unsigned int jobs_count,
                   unsigned int jobs_offset,unsigned int *global_count,unsigned int avg_degree){
    if(avg_degree <= 3){
        search_kernel_virtual_warp<<<BLK_NUMS,BLK_DIM>>>(query_pointers,data_pointers,c_pointers,
                                                         s_pointers,U,iter,jobs_count,
                                                         jobs_offset,global_count);
    }else{
        search_kernel<<<BLK_NUMS,BLK_DIM>>>(query_pointers,data_pointers,c_pointers,
                                            s_pointers,U,iter,jobs_count,
                                            jobs_offset,global_count);
    }
}

// the main search method
unsigned long long int search_mpi(string query_file,string data_file,int world_size, int rank,bool write_to_disk) {

    // load graph and allocate memory
    cout<<"start loading graph file from disk to memory..."<<endl;
    Graph query_graph(0,query_file);
    Graph data_graph(1,data_file);
    cout<<"graph loading complete..."<<endl;
    string q_base_name = base_name(query_file);
    string d_base_name = base_name(data_file);
    G_pointers query_pointers;
    G_pointers data_pointers;
    C_pointers c_pointers;
    S_pointers s_pointers;

    // create timing
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);

    // iters is the number of vertices
    unsigned int iters = query_graph.V;

    // allocate gpu memory
    malloc_graph_gpu_memory(query_graph,query_pointers);
    malloc_graph_gpu_memory(data_graph,data_pointers);
    malloc_query_constraints_gpu_memory(query_graph,c_pointers);

    // UNSURE - think this is what determines whether we use virtual warps or not
    if(data_graph.AVG_DEGREE <= 3){
        malloc_other_searching_gpu_memory(s_pointers,BLK_NUMS*WARPS_EACH_BLK*4,query_graph.V);
    }else{
        malloc_other_searching_gpu_memory(s_pointers,BLK_NUMS*WARPS_EACH_BLK,query_graph.V);
    }

    // UNSURE - 
    unsigned int *global_count;
    unsigned int results_count;
    chkerr(hipMalloc(&global_count,sizeof(unsigned int)));

    // allocates a buffer for each thread to recieve messages and work
    unsigned int *mpiCommBuffer = new unsigned int[8000000000];

    // each thread will try to recieve a message from all others, this will create a status object for all threads to threads, see rq_recv_msg
    mpi_irecv_all(rank); // open communication channels

    // initialize the global free list to all false, meaning no threads are free
    for (int i = 0; i < wsize; ++i) {
        global_free_list[i] = false;
    }

    // start timing
    hipEventRecord(event_start);

    // according to Brian finds hightest outdeg vertex and initializes candidates of it, initializes results table on gpu
    initialize_searching<<<108,512>>>(query_pointers.signatures,data_pointers.signatures,s_pointers.results_table,
                                     c_pointers.order_sqeuence,data_graph.V,s_pointers.lengths,world_size,rank);
    chkerr(hipDeviceSynchronize());

    // checks first whether there could be any candidates
    unsigned int *cans_array = new unsigned int[s_pointers.lengths[1]];
    unsigned int ini_count = s_pointers.lengths[1];
    if(ini_count == 0){
        return 0;
    }

    // copies candidates back from gpu to cpu
    chkerr(hipMemcpy(cans_array,s_pointers.results_table,ini_count*sizeof(unsigned int),hipMemcpyDeviceToHost));
    
    // UNSURE - shuffles the candidates, maybe we should have this
    shuffle_array(cans_array,ini_count);

    // duplicate initialization there is one shortly below???
    bool helpOthers = false;

    // UNSURE - 
    hipMemset(s_pointers.lengths,0,(iters+1)*sizeof(unsigned int));

    // UNSURE - think this is the number of block of work
    unsigned int trunk_size = 512;
    unsigned int num_trunks = (ini_count - 1)/trunk_size + 1;

    // UNSURE - for each block of work
    for(unsigned int l=0;l<num_trunks;++l){

        // UNSURE - what is iter?
        iter = 1;

        // set t_size as size of block normal for all but last block, should be if-else
        unsigned int t_size = trunk_size;
        if(l == num_trunks - 1){
            t_size = ini_count - l*trunk_size;
        }

        // initialize data for kernel
        // set all lengths to 0
        hipMemset(s_pointers.lengths,0,(iters+1)*sizeof(unsigned int));
        // UNSURE - 
        s_pointers.lengths[1] = t_size;
        // copy cans_array trunk to results table
        chkerr(hipMemcpy(s_pointers.results_table,&cans_array[l*trunk_size],
                          t_size*sizeof(unsigned int),hipMemcpyHostToDevice));

        // initialize help other to false
        helpOthers = false;

        // loop until all threads have no work remaining
        do {

            int taker;
            bool divided_work;

            // when we have gotten through a loop we have finished all work and gotten work from another thread so decode it and start helping
            // UNSURE - what is really the need of these conditions, they both seem to do the same thing, preventing this decoding on the first loop
            if (helpOthers && iter < iters) {


                hipMemset(s_pointers.lengths,0,(iters+1)*sizeof(unsigned int));
                iter = decode_com_buffer(mpiCommBuffer,s_pointers);
            }

            // for each task
            // UNSURE - understand that this is some breaking up of the work but have no clue what one iter actually is, unimportant
            for (;iter < iters; ++iter) {

                // UNSURE - 
                s_pointers.lengths[iter+1] = s_pointers.lengths[iter];
                hipMemset(global_count,0,sizeof(unsigned int));

                // UNSURE - 
                // reprents the amount of work the current iter will be
                unsigned int preCandidates = s_pointers.lengths[iter] - s_pointers.lengths[iter-1];
                
                // if current iter is a lot of work attempt to get help from another thread
                if(preCandidates > 100000){

                    // how many batches will the work will be partitioned into, one partition may be given to another for help
                    unsigned int miniBatchSize = preCandidates / 3;
                    
                    // perform the first batch of work
                    hipMemset(global_count,0,1*sizeof(unsigned int));
                    kernel_launch(query_pointers,data_pointers,c_pointers,
                                  s_pointers,data_graph.V,iter,miniBatchSize,
                                  0,global_count,data_graph.AVG_DEGREE);
                    chkerr(hipDeviceSynchronize());

                    // encode the data that might be sent
                    encode_com_buffer(mpiCommBuffer,s_pointers,iter,miniBatchSize);
                    
                    // attempt to give the work to another thread and indicate to all free thread that you need help if they cant respond immediately
                    divided_work = give_work_wrapper(grank, taker, mpiCommBuffer);
                    
                    // perform the third batch of work
                    hipMemset(global_count,0,1*sizeof(unsigned int));
                    kernel_launch(query_pointers,data_pointers,c_pointers,
                                  s_pointers,data_graph.V,iter,
                                  preCandidates-2*miniBatchSize,
                                  2*miniBatchSize,global_count,data_graph.AVG_DEGREE);
                    chkerr(hipDeviceSynchronize());
                    
                    // if we werent able to give work to another thread we must do it ourselves
                    if(!divided_work){

                        // perform the second batch of work
                        hipMemset(global_count,0,1*sizeof(unsigned int));
                        kernel_launch(query_pointers,data_pointers,c_pointers,s_pointers,data_graph.V,iter,
                                      miniBatchSize,miniBatchSize,global_count,data_graph.AVG_DEGREE);
                    }

                // there wasnt enough work to consider splitting it so just do it all our selves
                }else{

                    // perform all the work
                    hipMemset(global_count,0,1*sizeof(unsigned int));
                    kernel_launch(query_pointers,data_pointers,c_pointers,
                                  s_pointers,data_graph.V,iter,preCandidates,
                                  0,global_count,data_graph.AVG_DEGREE);
                }
                chkerr(hipDeviceSynchronize());

                // UNSURE - have no idea what this means, but it seems like we can break early if it occurs
                results_count = s_pointers.lengths[iter+1] - s_pointers.lengths[iter];
                if (results_count == 0) {
                    iter = iters;
                    break;
                }
            }

            // current thread is done with work so we cna now help others
            helpOthers = true;

        // returns the number of threads done working so will break when all work is done, other break from take_work would be when thread has taken work form another
        } while (wsize != take_work_wrap(rank, mpiCommBuffer));
    }

    // end timing
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_milli_sec = 0;
    hipEventElapsedTime(&time_milli_sec, event_start, event_stop);
    cout<<rank<<","<<d_base_name<<","<<q_base_name<<","<<time_milli_sec<<"ms,"<<s_pointers.final_count[0]<<endl;

    // write results to disk and return them
    if(write_to_disk){
        cout<<"start writting matching results to disk,ans.txt"<<endl;
        write_match_to_disk(s_pointers.indexes_pos[0],s_pointers.final_results_row_ptrs,query_graph.V,
                            query_graph.order_sequence,s_pointers.final_results_table);
        cout<<"finish writting matching results to disk,ans.txt"<<endl;
    }
    return s_pointers.final_count[0];
}

// the main method, just initialized MPI and calls the search method
int main(int argc, char *argv[]) {
    MPI_Init(&argc,&argv);

    // number of cpu threads
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD,&world_size);

    // current cpu threads rank
    int world_rank;
    MPI_Comm_rank(MPI_COMM_WORLD,&world_rank);

    std::string query_graph_file = argv[2];
    std::string data_graph_file = argv[1];
    bool write_to_disk = false;

    // main search method
    unsigned long long int result_len = search_mpi(query_graph_file,data_graph_file,world_size,world_rank,write_to_disk);

    // end mpi
    MPI_Finalize();
    return 0;
}
